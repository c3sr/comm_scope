#include "hip/hip_runtime.h"
#include <sstream>

#include "scope/scope.hpp"

#include "../args.hpp"

#define NAME "Comm_3d_kernel3D_push"

__global__ void Comm_3d_kernel3D_push_kernel(void *__restrict__ dst,
                                             const void *__restrict__ src,
                                             const hipExtent allocExtent,
                                             const hipExtent copyExtent,
                                             const size_t elemSize) {

  const unsigned int tz = blockDim.z * blockIdx.z + threadIdx.z;
  const unsigned int ty = blockDim.y * blockIdx.y + threadIdx.y;
  const unsigned int tx = blockDim.x * blockIdx.x + threadIdx.x;

  for (unsigned int zi = tz; zi < copyExtent.depth;
       zi += blockDim.z * gridDim.z) {
    for (unsigned int yi = ty; yi < copyExtent.height;
         yi += blockDim.y * gridDim.y) {
      for (unsigned int xi = tx; xi < copyExtent.width;
           xi += blockDim.x * gridDim.x) {
        unsigned int ii = zi * allocExtent.height * allocExtent.width +
                          yi * allocExtent.width + xi;
        if (4 == elemSize) {
          uint32_t *pDst = reinterpret_cast<uint32_t *>(dst);
          const uint32_t *pSrc = reinterpret_cast<const uint32_t *>(src);
          uint32_t v = pSrc[ii];
          pDst[ii] = v;
        } else if (8 == elemSize) {
          uint64_t *pDst = reinterpret_cast<uint64_t *>(dst);
          const uint64_t *pSrc = reinterpret_cast<const uint64_t *>(src);
          pDst[ii] = pSrc[ii];
        } else {
          char *pDst = reinterpret_cast<char *>(dst);
          const char *pSrc = reinterpret_cast<const char *>(src);
          memcpy(&pDst[ii * elemSize], &pSrc[ii * elemSize], elemSize);
        }
      }
    }
  }
}

inline int64_t nextPowerOfTwo(int64_t x) {
  x--;
  x |= x >> 1;
  x |= x >> 2;
  x |= x >> 4;
  x |= x >> 8;
  x |= x >> 16;
  x |= x >> 32;
  x++;
  return x;
}

inline dim3 make_block_dim(const hipExtent extent, int64_t threads) {
  assert(threads <= 1024);
  dim3 ret;
  ret.x = std::min(threads, nextPowerOfTwo(extent.width));
  threads /= ret.x;
  ret.y = std::min(threads, nextPowerOfTwo(extent.height));
  threads /= ret.y;
  ret.z = std::min(threads, nextPowerOfTwo(extent.depth));

  // if z is too big, push down into y
  if (ret.z > 64) {
    ret.y *= (ret.z / 64);
    ret.z = 64;
  }

  assert(ret.x <= 1024);
  assert(ret.y <= 1024);
  assert(ret.z <= 64); // maximum
  assert(ret.x * ret.y * ret.z <= 1024);
  return ret;
}

auto Comm_3d_kernel3D_push = [](benchmark::State &state, const int gpu0,
                                const int gpu1) {

#if defined(SCOPE_USE_NVTX)
  {
    std::stringstream name;
    name << NAME << "/" << gpu0 << "/" << gpu1 << "/" << state.range(0) << "/"
         << state.range(1) << "/" << state.range(2);
    nvtxRangePush(name.str().c_str());
  }
#endif

  OR_SKIP_AND_RETURN(scope::cuda_reset_device(gpu0),
                     NAME " failed to reset CUDA device");
  OR_SKIP_AND_RETURN(scope::cuda_reset_device(gpu1),
                     NAME " failed to reset CUDA device");

  // create stream on src gpu (push)
  OR_SKIP_AND_RETURN(hipSetDevice(gpu0), NAME "failed to create stream");
  hipStream_t stream = nullptr;
  OR_SKIP_AND_RETURN(hipStreamCreate(&stream), NAME "failed to create stream");

  // Start and stop events on src gpu (push)
  hipEvent_t start = nullptr;
  hipEvent_t stop = nullptr;
  OR_SKIP_AND_RETURN(hipEventCreate(&start), NAME " failed to create event");
  OR_SKIP_AND_RETURN(hipEventCreate(&stop), NAME " failed to create event");

  // target size to transfer
  hipExtent copyExt;
  copyExt.width = static_cast<size_t>(state.range(0));
  copyExt.height = static_cast<size_t>(state.range(1));
  copyExt.depth = static_cast<size_t>(state.range(2));
  const size_t copyBytes = copyExt.width * copyExt.height * copyExt.depth;

  // properties of the allocation
  hipExtent allocExt;
  allocExt.width  = 768*4;  // how many bytes in a row
  allocExt.height = 768; // how many rows in a plane
  allocExt.depth  = 768;

  hipPitchedPtr src, dst;

  // allocate on gpu0 and enable peer access
  OR_SKIP_AND_RETURN(hipSetDevice(gpu0), NAME "failed to set device");
  OR_SKIP_AND_RETURN(hipMalloc3D(&src, allocExt),
                     NAME " failed to perform hipMalloc3D");
  allocExt.width = src.pitch;
  OR_SKIP_AND_RETURN(hipMemset3D(src, 0, allocExt),
                     NAME " failed to perform src hipMemset3D");
  if (gpu0 != gpu1) {
    hipError_t err = hipDeviceEnablePeerAccess(gpu1, 0);
    if (hipSuccess != err && hipErrorPeerAccessAlreadyEnabled != err) {
      state.SkipWithError(NAME " failed to ensure peer access");
    }
  }

  // allocate on gpu1 and enable peer access
  OR_SKIP_AND_RETURN(hipSetDevice(gpu1), NAME "failed to set device");
  OR_SKIP_AND_RETURN(hipMalloc3D(&dst, allocExt),
                     NAME " failed to perform hipMalloc3D");
  OR_SKIP_AND_RETURN(hipMemset3D(dst, 0, allocExt),
                     NAME " failed to perform src hipMemset3D");
  if (gpu0 != gpu1) {
    hipError_t err = hipDeviceEnablePeerAccess(gpu0, 0);
    if (hipSuccess != err && hipErrorPeerAccessAlreadyEnabled != err) {
      state.SkipWithError(NAME " failed to ensure peer access");
    }
  }

  // 4 bytes per thread
  size_t elemSize = 4;
  // convert alloc and copy extent to be in terms of elemSize chunks
  assert(allocExt.width % elemSize == 0);
  allocExt.width /= elemSize;
  assert(copyExt.width % elemSize == 0);
  copyExt.width /= elemSize;

  dim3 blockDim = make_block_dim(copyExt, 512);
  dim3 gridDim;
  gridDim.x = (copyExt.width + blockDim.x - 1) / blockDim.x;
  gridDim.y = (copyExt.height + blockDim.y - 1) / blockDim.y;
  gridDim.z = (copyExt.depth + blockDim.z - 1) / blockDim.z;

  // push, so run kernel on source device
  OR_SKIP_AND_RETURN(hipSetDevice(gpu0), NAME " unable to set push device");

  for (auto _ : state) {
    // Start copy
    OR_SKIP_AND_BREAK(hipEventRecord(start, stream),
                      NAME " failed to record start event");

    Comm_3d_kernel3D_push_kernel<<<gridDim, blockDim, 0, stream>>>(
        dst.ptr, src.ptr, allocExt, copyExt, elemSize);
    OR_SKIP_AND_BREAK(hipGetLastError(), "kernel");

    OR_SKIP_AND_BREAK(hipEventRecord(stop, stream),
                      NAME " failed to record stop event");

    // Wait for all copies to finish
    OR_SKIP_AND_BREAK(hipEventSynchronize(stop),
                      NAME " failed to synchronize");

    // Get the transfer time
    float millis;
    OR_SKIP_AND_BREAK(hipEventElapsedTime(&millis, start, stop),
                      NAME " failed to compute elapsed tiume");
    state.SetIterationTime(millis / 1000);
  }

  state.SetBytesProcessed(int64_t(state.iterations()) * int64_t(copyBytes));
  state.counters["bytes"] = copyBytes;
  state.counters["gpu0"] = gpu0;
  state.counters["gpu1"] = gpu1;
  state.counters["dbx"] = blockDim.x;
  state.counters["dby"] = blockDim.y;
  state.counters["dbz"] = blockDim.z;
  state.counters["dgx"] = gridDim.x;
  state.counters["dgy"] = gridDim.y;
  state.counters["dgz"] = gridDim.x;

  OR_SKIP_AND_RETURN(hipEventDestroy(start), "hipEventDestroy");
  OR_SKIP_AND_RETURN(hipEventDestroy(stop), "hipEventDestroy");
  OR_SKIP_AND_RETURN(hipStreamDestroy(stream), "hipStreamDestroy");
  OR_SKIP_AND_RETURN(hipFree(src.ptr), "hipFree");
  OR_SKIP_AND_RETURN(hipFree(dst.ptr), "hipFree");

#if defined(SCOPE_USE_NVTX)
  nvtxRangePop();
#endif
};

static void registerer() {
  std::string name;
  const std::vector<MemorySpace> cudaSpaces = scope::system::memory_spaces(MemorySpace::Kind::cuda_device);

  for (const auto &space0 : cudaSpaces) {
    for (const auto &space1 : cudaSpaces) {

      auto gpu0 = space0.device_id();
      auto gpu1 = space1.device_id();
      int ok1, ok2;
      if (!PRINT_IF_ERROR(hipDeviceCanAccessPeer(&ok1, gpu0, gpu1)) &&
          !PRINT_IF_ERROR(hipDeviceCanAccessPeer(&ok2, gpu1, gpu0))) {
        if ((ok1 && ok2) || gpu0 == gpu1) {
          name = std::string(NAME) + "/" + std::to_string(gpu0) + "/" +
                 std::to_string(gpu1);
          benchmark::RegisterBenchmark(name.c_str(), Comm_3d_kernel3D_push,
                                       gpu0, gpu1)
              ->ASTAROTH_ARGS()
              ->UseManualTime();
        }
      }
    }
  }
}

SCOPE_AFTER_INIT(registerer, NAME);
