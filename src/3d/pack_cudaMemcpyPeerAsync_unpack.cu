#include "hip/hip_runtime.h"
#include <sstream>

#include "scope/scope.hpp"

#include "args.hpp"

#define NAME "Comm_3d_pack_cudaMemcpyPeer_unpack"

__global__ void Comm_3d_pack_cudaMemcpyPeer_unpack_pack_kernel(
    void *__restrict__ dst, const void *__restrict__ src,
    const hipExtent allocExtent, // in elements
    const hipExtent copyExtent,  // in elements
    const size_t elemSize) {

  const unsigned int tz = blockDim.z * blockIdx.z + threadIdx.z;
  const unsigned int ty = blockDim.y * blockIdx.y + threadIdx.y;
  const unsigned int tx = blockDim.x * blockIdx.x + threadIdx.x;

  for (unsigned int zi = tz; zi < copyExtent.depth;
       zi += blockDim.z * gridDim.z) {
    for (unsigned int yi = ty; yi < copyExtent.height;
         yi += blockDim.y * gridDim.y) {
      for (unsigned int xi = tx; xi < copyExtent.width;
           xi += blockDim.x * gridDim.x) {
        unsigned int ii = zi * allocExtent.height * allocExtent.width +
                          yi * allocExtent.width + xi;
        unsigned int oi = zi * copyExtent.height * copyExtent.width +
                          yi * copyExtent.width + xi;
        if (4 == elemSize) {
          uint32_t *pDst = reinterpret_cast<uint32_t *>(dst);
          const uint32_t *pSrc = reinterpret_cast<const uint32_t *>(src);
          uint32_t v = pSrc[ii];
          pDst[oi] = v;
        } else if (8 == elemSize) {
          uint64_t *pDst = reinterpret_cast<uint64_t *>(dst);
          const uint64_t *pSrc = reinterpret_cast<const uint64_t *>(src);
          pDst[oi] = pSrc[ii];
        } else {
          char *pDst = reinterpret_cast<char *>(dst);
          const char *pSrc = reinterpret_cast<const char *>(src);
          memcpy(&pDst[oi * elemSize], &pSrc[ii * elemSize], elemSize);
        }
      }
    }
  }
}

__global__ void Comm_3d_pack_cudaMemcpyPeer_unpack_unpack_kernel(
    void *__restrict__ dst, const void *__restrict__ src,
    const hipExtent allocExtent, const hipExtent copyExtent,
    const size_t elemSize) {

  const unsigned int tz = blockDim.z * blockIdx.z + threadIdx.z;
  const unsigned int ty = blockDim.y * blockIdx.y + threadIdx.y;
  const unsigned int tx = blockDim.x * blockIdx.x + threadIdx.x;

  for (unsigned int zi = tz; zi < copyExtent.depth;
       zi += blockDim.z * gridDim.z) {
    for (unsigned int yi = ty; yi < copyExtent.height;
         yi += blockDim.y * gridDim.y) {
      for (unsigned int xi = tx; xi < copyExtent.width;
           xi += blockDim.x * gridDim.x) {
        unsigned int oi = zi * allocExtent.height * allocExtent.width +
                          yi * allocExtent.width + xi;
        unsigned int ii = zi * copyExtent.height * copyExtent.width +
                          yi * copyExtent.width + xi;
        if (4 == elemSize) {
          uint32_t *pDst = reinterpret_cast<uint32_t *>(dst);
          const uint32_t *pSrc = reinterpret_cast<const uint32_t *>(src);
          uint32_t v = pSrc[ii];
          pDst[oi] = v;
        } else if (8 == elemSize) {
          uint64_t *pDst = reinterpret_cast<uint64_t *>(dst);
          const uint64_t *pSrc = reinterpret_cast<const uint64_t *>(src);
          pDst[oi] = pSrc[ii];
        } else {
          char *pDst = reinterpret_cast<char *>(dst);
          const char *pSrc = reinterpret_cast<const char *>(src);
          memcpy(&pDst[oi * elemSize], &pSrc[ii * elemSize], elemSize);
        }
      }
    }
  }
}

inline int64_t nextPowerOfTwo(int64_t x) {
  x--;
  x |= x >> 1;
  x |= x >> 2;
  x |= x >> 4;
  x |= x >> 8;
  x |= x >> 16;
  x |= x >> 32;
  x++;
  return x;
}

inline dim3 make_block_dim(const hipExtent extent, int64_t threads) {
  assert(threads <= 1024);
  dim3 ret;
  ret.x = std::min(threads, nextPowerOfTwo(extent.width));
  threads /= ret.x;
  ret.y = std::min(threads, nextPowerOfTwo(extent.height));
  threads /= ret.y;
  ret.z = std::min(threads, nextPowerOfTwo(extent.depth));

  // if z is too big, push down into y
  if (ret.z > 64) {
    ret.y *= (ret.z / 64);
    ret.z = 64;
  }

  assert(ret.x <= 1024);
  assert(ret.y <= 1024);
  assert(ret.z <= 64); // maximum
  assert(ret.x * ret.y * ret.z <= 1024);
  return ret;
}

auto Comm_3d_pack_cudaMemcpyPeer_unpack = [](benchmark::State &state,
                                             const int gpu0, const int gpu1) {

#if defined(SCOPE_USE_NVTX)
  {
    std::stringstream name;
    name << NAME << "/" << gpu0 << "/" << gpu1 << "/" << state.range(0) << "/"
         << state.range(1) << "/" << state.range(2);
    nvtxRangePush(name.str().c_str());
  }
#endif

  OR_SKIP_AND_RETURN(cuda_reset_device(gpu0),
                     NAME " failed to reset CUDA device");
  OR_SKIP_AND_RETURN(cuda_reset_device(gpu1),
                     NAME " failed to reset CUDA device");

  // create stream on src gpu for pack + copy
  OR_SKIP_AND_RETURN(hipSetDevice(gpu0), NAME " failed to set device");
  hipStream_t srcStream = nullptr;
  OR_SKIP_AND_RETURN(hipStreamCreate(&srcStream),
                     NAME " failed to create source stream");

  // create a stream on the dst gpu for unpack
  OR_SKIP_AND_RETURN(hipSetDevice(gpu1), NAME "failed to set device");
  hipStream_t dstStream = nullptr;
  OR_SKIP_AND_RETURN(hipStreamCreate(&dstStream),
                     NAME "failed to create dst stream");

  // Start and stop events on src gpu
  OR_SKIP_AND_RETURN(hipSetDevice(gpu0), NAME "failed to create stream");
  hipEvent_t start = nullptr;
  hipEvent_t stop = nullptr;
  OR_SKIP_AND_RETURN(hipEventCreate(&start), NAME " failed to create event");
  OR_SKIP_AND_RETURN(hipEventCreate(&stop), NAME " failed to create event");

  // event to serialize unpack after copy
  hipEvent_t copyDone = nullptr;
  OR_SKIP_AND_RETURN(hipEventCreate(&copyDone),
                     NAME " failed to create event");

  // event to serialize stop after unpack
  OR_SKIP_AND_RETURN(hipSetDevice(gpu1), "hipSetDevice");
  hipEvent_t unpackDone = nullptr;
  OR_SKIP_AND_RETURN(hipEventCreate(&unpackDone),
                     " failed to created unpackDone event");

  // target size to transfer
  hipExtent copyExt;
  copyExt.width = static_cast<size_t>(state.range(0));
  copyExt.height = static_cast<size_t>(state.range(1));
  copyExt.depth = static_cast<size_t>(state.range(2));
  const size_t copyBytes = copyExt.width * copyExt.height * copyExt.depth;

  // properties of the allocation
  hipExtent allocExt;
  allocExt.width = 768 * 4; // how many bytes in a row
  allocExt.height = 768;    // how many rows in a plane
  allocExt.depth = 768;

  // 3D regions
  hipPitchedPtr src, dst;

  // 1D pack/unpack buffers
  void *srcBuf, *dstBuf;

  // allocate on gpu0 and enable peer access
  OR_SKIP_AND_RETURN(hipSetDevice(gpu0), NAME "failed to set device");
  OR_SKIP_AND_RETURN(hipMalloc3D(&src, allocExt),
                     NAME " failed to perform hipMalloc3D");
  allocExt.width = src.pitch;
  OR_SKIP_AND_RETURN(hipMalloc(&srcBuf, copyBytes),
                     NAME " failed to alloc flat src buffer");
  if (gpu0 != gpu1) {
    hipError_t err = hipDeviceEnablePeerAccess(gpu1, 0);
    if (hipSuccess != err && hipErrorPeerAccessAlreadyEnabled != err) {
      state.SkipWithError(NAME " failed to ensure peer access");
    }
  }

  // allocate on gpu1 and enable peer access
  OR_SKIP_AND_RETURN(hipSetDevice(gpu1), NAME "failed to set device");
  OR_SKIP_AND_RETURN(hipMalloc3D(&dst, allocExt),
                     NAME " failed to perform hipMalloc3D");
  OR_SKIP_AND_RETURN(hipMalloc(&dstBuf, copyBytes),
                     NAME " failed to alloc flat dst buffer");
  if (gpu0 != gpu1) {
    hipError_t err = hipDeviceEnablePeerAccess(gpu0, 0);
    if (hipSuccess != err && hipErrorPeerAccessAlreadyEnabled != err) {
      state.SkipWithError(NAME " failed to ensure peer access");
    }
  }

  // 4 bytes per thread
  size_t elemSize = 4;
  // convert alloc and copy extent to be in terms of elemSize chunks
  assert(allocExt.width % elemSize == 0);
  allocExt.width /= elemSize;
  assert(copyExt.width % elemSize == 0);
  copyExt.width /= elemSize;

  dim3 blockDim = make_block_dim(copyExt, 512);
  dim3 gridDim;
  gridDim.x = (copyExt.width + blockDim.x - 1) / blockDim.x;
  gridDim.y = (copyExt.height + blockDim.y - 1) / blockDim.y;
  gridDim.z = (copyExt.depth + blockDim.z - 1) / blockDim.z;

  for (auto _ : state) {
    OR_SKIP_AND_BREAK(hipSetDevice(gpu0), "hipSetDevice(gpu0)");

    // Record start
    OR_SKIP_AND_BREAK(hipEventRecord(start, srcStream),
                      "failed to record start event");

    // pack on source
    Comm_3d_pack_cudaMemcpyPeer_unpack_pack_kernel<<<gridDim, blockDim, 0,
                                                     srcStream>>>(
        srcBuf, src.ptr, allocExt, copyExt, elemSize);
    OR_SKIP_AND_BREAK(hipGetLastError(), "pack kernel");

    // copy
    OR_SKIP_AND_BREAK(
        hipMemcpyPeerAsync(dstBuf, gpu1, srcBuf, gpu0, copyBytes, srcStream),
        "hipMemcpyPeerAsync");

    // block dst stream until copy finished
    OR_SKIP_AND_BREAK(hipEventRecord(copyDone, srcStream), "copyDone");
    OR_SKIP_AND_BREAK(hipStreamWaitEvent(dstStream, copyDone, 0 /*must be 0*/),
                      "hipStreamWaitEvent");

    // unpack on dst
    OR_SKIP_AND_BREAK(hipSetDevice(gpu1), "hipSetDevice(gpu1)");
    Comm_3d_pack_cudaMemcpyPeer_unpack_unpack_kernel<<<gridDim, blockDim, 0,
                                                       dstStream>>>(
        dst.ptr, dstBuf, allocExt, copyExt, elemSize);
    OR_SKIP_AND_BREAK(hipGetLastError(), "unpack kernel");

    // record unpack done
    OR_SKIP_AND_BREAK(hipEventRecord(unpackDone, dstStream),
                      "hipEventRecord(unpackDone, dstStream)");

    // record all operations done
    OR_SKIP_AND_BREAK(
        hipStreamWaitEvent(srcStream, unpackDone, 0 /*must be 0*/),
        "wait for unpackDone");
    OR_SKIP_AND_BREAK(hipEventRecord(stop, srcStream),
                      NAME " failed to record stop event");

    // Wait for all copies to finish
    OR_SKIP_AND_BREAK(hipEventSynchronize(stop), "hipEventSynchronize(stop)");

    // Get the transfer time
    float millis;
    OR_SKIP_AND_BREAK(hipEventElapsedTime(&millis, start, stop),
                      NAME " failed to compute elapsed time");
    state.SetIterationTime(millis / 1000);
  }

  state.SetBytesProcessed(int64_t(state.iterations()) * int64_t(copyBytes));
  state.counters["bytes"] = copyBytes;
  state.counters["gpu0"] = gpu0;
  state.counters["gpu1"] = gpu1;
  state.counters["dbx"] = blockDim.x;
  state.counters["dby"] = blockDim.y;
  state.counters["dbz"] = blockDim.z;
  state.counters["dgx"] = gridDim.x;
  state.counters["dgy"] = gridDim.y;
  state.counters["dgz"] = gridDim.x;

  OR_SKIP_AND_RETURN(hipFree(src.ptr), "hipFree");
  OR_SKIP_AND_RETURN(hipFree(dst.ptr), "hipFree");
  OR_SKIP_AND_RETURN(hipFree(srcBuf), "hipFree");
  OR_SKIP_AND_RETURN(hipFree(dstBuf), "hipFree");
  OR_SKIP_AND_RETURN(hipStreamDestroy(srcStream), "hipStreamDestroy");
  OR_SKIP_AND_RETURN(hipStreamDestroy(dstStream), "hipStreamDestroy");
  OR_SKIP_AND_RETURN(hipEventDestroy(start), "hipEventDestroy");
  OR_SKIP_AND_RETURN(hipEventDestroy(stop), "hipEventDestroy");
  OR_SKIP_AND_RETURN(hipEventDestroy(copyDone), "hipEventDestroy");
  OR_SKIP_AND_RETURN(hipEventDestroy(unpackDone), "hipEventDestroy");

#if defined(SCOPE_USE_NVTX)
  nvtxRangePop();
#endif
};

static void registerer() {
  std::string name;

  const std::vector<MemorySpace> cudaSpaces =
      scope::system::memory_spaces(MemorySpace::Kind::cuda_device);

  for (const auto &space0 : cudaSpaces) {
    for (const auto &space1 : cudaSpaces) {

      auto id0 = space0.device_id();
      auto id1 = space1.device_id();
      int ok1, ok2;
      if (!PRINT_IF_ERROR(hipDeviceCanAccessPeer(&ok1, id0, id1)) &&
          !PRINT_IF_ERROR(hipDeviceCanAccessPeer(&ok2, id1, id0))) {
        if ((ok1 && ok2) || id0 == id1) {
          name = std::string(NAME) + "/" + std::to_string(id0) + "/" +
                 std::to_string(id1);
          benchmark::RegisterBenchmark(
              name.c_str(), Comm_3d_pack_cudaMemcpyPeer_unpack, id0, id1)
              ->ASTAROTH_ARGS()
              ->UseManualTime();
        }
      }
    }
  }
}

SCOPE_AFTER_INIT(registerer, NAME);
