
#include <hip/hip_runtime.h>
#if __CUDACC_VER_MAJOR__ >= 8

#include "scope/scope.hpp"

#include "args.hpp"

#define NAME "Comm_UM_Prefetch_GPUToHost"

auto Comm_UM_Prefetch_GPUToHost = [](benchmark::State &state, const int numa_id,
                                     const int cuda_id) {
  const auto bytes = 1ULL << static_cast<size_t>(state.range(0));

  numa::bind_node(numa_id);

  if (PRINT_IF_ERROR(scope::cuda_reset_device(cuda_id))) {
    state.SkipWithError(NAME " failed to reset device");
    return;
  }

  if (PRINT_IF_ERROR(cudaSetDevice(cuda_id))) {
    state.SkipWithError(NAME " failed to set CUDA device");
    return;
  }

  char *ptr = nullptr;
  if (PRINT_IF_ERROR(cudaMallocManaged(&ptr, bytes))) {
    state.SkipWithError(NAME " failed to perform cudaMallocManaged");
    return;
  }
  defer(cudaFree(ptr));

  if (PRINT_IF_ERROR(cudaMemset(ptr, 0, bytes))) {
    state.SkipWithError(NAME " failed to perform cudaMemset");
    return;
  }

  cudaEvent_t start, stop;
  if (PRINT_IF_ERROR(cudaEventCreate(&start))) {
    state.SkipWithError(NAME " failed to create start event");
    return;
  }
  defer(cudaEventDestroy(start));

  if (PRINT_IF_ERROR(cudaEventCreate(&stop))) {
    state.SkipWithError(NAME " failed to create end event");
    return;
  }
  defer(cudaEventDestroy(stop));

  for (auto _ : state) {
    if (PRINT_IF_ERROR(cudaMemPrefetchAsync(ptr, bytes, cuda_id))) {
      state.SkipWithError(NAME " failed to prefetch to src");
      return;
    }
    if (PRINT_IF_ERROR(cudaDeviceSynchronize())) {
      state.SkipWithError(NAME " failed to synchronize");
      return;
    }
    cudaEventRecord(start);
    if (PRINT_IF_ERROR(cudaMemPrefetchAsync(ptr, bytes, cudaCpuDeviceId))) {
      state.SkipWithError(NAME " failed to move data to dst");
      return;
    }
    cudaEventRecord(stop);
    cudaEventSynchronize(stop);

    float millis = 0;
    if (PRINT_IF_ERROR(cudaEventElapsedTime(&millis, start, stop))) {
      state.SkipWithError(NAME " failed to get elapsed time");
      break;
    }
    state.SetIterationTime(millis / 1000);
  }

  state.SetBytesProcessed(int64_t(state.iterations()) * int64_t(bytes));
  state.counters["bytes"] = bytes;
  state.counters["cuda_id"] = cuda_id;
  state.counters["numa_id"] = numa_id;

  // reset to run on any node
  numa::bind_node(-1);
};

static void registerer() {
  for (int cuda_id : scope::system::cuda_devices()) {
    for (auto numa_id : numa::mems()) {
      std::string name = std::string(NAME) + "/" + std::to_string(numa_id) +
                         "/" + std::to_string(cuda_id);
      benchmark::RegisterBenchmark(name.c_str(), Comm_UM_Prefetch_GPUToHost,
                                   numa_id, cuda_id)
          ->SMALL_ARGS()
          ->UseManualTime();
    }
  }
}

SCOPE_AFTER_INIT(registerer, NAME);

#endif // __CUDACC_VER_MAJOR__ >= 8