/* zero-copy write workload
 */

#include "scope/scope.hpp"

#include "args.hpp"
#include "kernels.hu"

#define NAME "Comm_ZeroCopy_GPUToHost"

auto Comm_ZeroCopy_GPUToHost = [](benchmark::State &state, const int src_numa,
                                  const int dst_cuda) {
  const size_t pageSize = page_size();

  const auto bytes = 1ULL << static_cast<size_t>(state.range(0));

  numa::ScopedBind binder(src_numa);

  OR_SKIP_AND_RETURN(scope::cuda_reset_device(dst_cuda), "");
  OR_SKIP_AND_RETURN(hipSetDevice(dst_cuda), "");

  void *ptr = aligned_alloc(pageSize, bytes);
  defer(free(ptr));
  if (!ptr && bytes) {
    state.SkipWithError(NAME " failed to allocate host memory");
    return;
  }
  std::memset(ptr, 0, bytes);

  OR_SKIP_AND_RETURN(hipHostRegister(ptr, bytes, hipHostRegisterMapped), "");
  defer(hipHostUnregister(ptr));

  // get a valid device pointer
  void *dptr;
  hipDeviceProp_t prop;
  OR_SKIP_AND_RETURN(hipGetDeviceProperties(&prop, dst_cuda), "");

#if __CUDACC_VER_MAJOR__ >= 9
  if (prop.canUseHostPointerForRegisteredMem) {
#else
  if (false) {
#endif
    dptr = ptr;
  } else {
    OR_SKIP_AND_RETURN(hipHostGetDevicePointer(&dptr, ptr, 0), "");
  }

  hipEvent_t start, stop;
  OR_SKIP_AND_RETURN(hipEventCreate(&start), "");
  defer(hipEventDestroy(start));
  OR_SKIP_AND_RETURN(hipEventCreate(&stop), "");
  defer(hipEventDestroy(stop));

  for (auto _ : state) {

    OR_SKIP_AND_BREAK(hipEventRecord(start), "");
    constexpr unsigned GD = 256;
    constexpr unsigned BD = 256;
    gpu_write<BD><<<GD, BD>>>((int32_t *)dptr, bytes);

    OR_SKIP_AND_BREAK(hipEventRecord(stop), "");
    OR_SKIP_AND_BREAK(hipEventSynchronize(stop), "");

    float millis = 0;
    OR_SKIP_AND_BREAK(hipEventElapsedTime(&millis, start, stop), "");
    state.SetIterationTime(millis / 1000);
  }

  state.SetBytesProcessed(int64_t(state.iterations()) * int64_t(bytes));
  state.counters["bytes"] = bytes;
  state.counters["src_numa"] = src_numa;
  state.counters["dst_cuda"] = dst_cuda;
};

static void registerer() {

  for (int cuda_id : scope::system::cuda_devices()) {
    for (auto numa_id : numa::mems()) {
      std::string name = std::string(NAME) + "/" + std::to_string(numa_id) +
                         "/" + std::to_string(cuda_id);
      benchmark::RegisterBenchmark(name.c_str(), Comm_ZeroCopy_GPUToHost,
                                   numa_id, cuda_id)
          ->ARGS()
          ->UseManualTime();
    }
  }
}

SCOPE_AFTER_INIT(registerer, NAME);
