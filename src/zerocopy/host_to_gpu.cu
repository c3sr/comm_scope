#include "scope/scope.hpp"

#include "args.hpp"
#include "kernels.hu"

#define NAME "Comm_ZeroCopy_HostToGPU"

enum class ShouldFlush { No, Yes };

const char * to_string(ShouldFlush flush) {
  switch (flush) {
    case ShouldFlush::No: return "";
    case ShouldFlush::Yes: return "_flush";
  }
  exit(EXIT_FAILURE);
}

auto Comm_ZeroCopy_HostToGPU = [](benchmark::State &state, const int src_numa,
                                  const int dst_cuda, const ShouldFlush flush) {
  const size_t pageSize = page_size();

  const auto bytes = 1ULL << static_cast<size_t>(state.range(0));

  numa::ScopedBind binder(src_numa);

  OR_SKIP_AND_RETURN(cuda_reset_device(dst_cuda), "");
  OR_SKIP_AND_RETURN(hipSetDevice(dst_cuda), "");

  void *ptr = aligned_alloc(pageSize, bytes);
  defer(free(ptr));
  if (!ptr && bytes) {
    state.SkipWithError(NAME " failed to allocate host memory");
    return;
  }
  std::memset(ptr, 0xDEADBEEF, bytes);

  OR_SKIP_AND_RETURN(hipHostRegister(ptr, bytes, hipHostRegisterMapped), "");
  defer(hipHostUnregister(ptr));

  // get a valid device pointer
  void *dptr;
  hipDeviceProp_t prop;
  OR_SKIP_AND_RETURN(hipGetDeviceProperties(&prop, dst_cuda), "");

#if __CUDACC_VER_MAJOR__ >= 9
  if (prop.canUseHostPointerForRegisteredMem) {
#else
  if (false) {
#endif
    dptr = ptr;
  } else {
    OR_SKIP_AND_RETURN(hipHostGetDevicePointer(&dptr, ptr, 0), "");
  }

  hipEvent_t start, stop;
  OR_SKIP_AND_RETURN(hipEventCreate(&start), "");
  defer(hipEventDestroy(start));
  OR_SKIP_AND_RETURN(hipEventCreate(&stop), "");
  defer(hipEventDestroy(stop));

  for (auto _ : state) {

    if (ShouldFlush::Yes == flush) {
      flush_all(ptr, bytes);
    }

    OR_SKIP_AND_BREAK(hipEventRecord(start), "");
    constexpr unsigned GD = 256;
    constexpr unsigned BD = 256;
    gpu_read<BD><<<GD, BD>>>((int32_t *)dptr, (int32_t *)nullptr, bytes);

    OR_SKIP_AND_BREAK(hipEventRecord(stop), "");
    OR_SKIP_AND_BREAK(hipEventSynchronize(stop), "");

    float millis = 0;
    OR_SKIP_AND_BREAK(hipEventElapsedTime(&millis, start, stop), "");
    state.SetIterationTime(millis / 1000);
  }

  state.SetBytesProcessed(int64_t(state.iterations()) * int64_t(bytes));
  state.counters["bytes"] = bytes;
  state.counters["src_numa"] = src_numa;
  state.counters["dst_cuda"] = dst_cuda;
};

static void registerer() {
  for (auto flush : {ShouldFlush::No, ShouldFlush::Yes}) {
    for (auto cuda_id : unique_cuda_device_ids()) {
      for (auto numa_id : numa::mems()) {
        std::string name = std::string(NAME) + to_string(flush) + "/" + std::to_string(numa_id) +
                           "/" + std::to_string(cuda_id);
        benchmark::RegisterBenchmark(name.c_str(), Comm_ZeroCopy_HostToGPU,
                                     numa_id, cuda_id, flush)
            ->ARGS()
            ->UseManualTime();
      }
    }
  }
}

SCOPE_AFTER_INIT(registerer, NAME);
