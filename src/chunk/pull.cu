#include "hip/hip_runtime.h"
#include <sstream>

#include "scope/scope.hpp"

#include "args.hpp"

#define NAME "Comm_chunk_pull"

typedef int read_t;

/* chunkFill should be lte 32
*/
static __global__ void Comm_chunk_pull_kernel(read_t *__restrict__ src,
                                              const int chunkSize,
                                              const int chunkFill,
                                              const int n, // number of chunks
                                              read_t * __restrict__ flag) {
  const int li = threadIdx.x % 32; // lane index
  const int wi = threadIdx.x / 32; // warp index
  const int bd = blockDim.x / 32;  // dimension of block in warps

  // assign one warp to each chunk
  for (int i = bd *blockIdx.x + wi; i < n; i += gridDim.x * bd) {
    if (li < chunkFill) {
      read_t t;
      do_not_optimize(t = src[i * chunkSize + li]);
      if (flag) {
        *flag = t;
      }
    }
  }
}

auto Comm_chunk_pull = [](benchmark::State &state, const int gpu0,
                          const int gpu1) {

#if defined(SCOPE_USE_NVTX)
  {
    std::stringstream name;
    name << NAME << "/" << gpu0 << "/" << gpu1 << "/" << state.range(0) << "/"
         << state.range(1);
    nvtxRangePush(name.str().c_str());
  }
#endif

  // in read_t, not bytes
  const int chunkSize = state.range(0);
  const int chunkFill = state.range(1);

  OR_SKIP_AND_RETURN(scope::cuda_reset_device(gpu0),
                     NAME " failed to reset CUDA device");
  OR_SKIP_AND_RETURN(scope::cuda_reset_device(gpu1),
                     NAME " failed to reset CUDA device");

  // create stream on dst gpu (pull)
  OR_SKIP_AND_RETURN(hipSetDevice(gpu1), NAME "failed to create stream");
  hipStream_t stream = nullptr;
  OR_SKIP_AND_RETURN(hipStreamCreate(&stream), NAME "failed to create stream");

  // Start and stop events on dst gpu (pull)
  hipEvent_t start = nullptr;
  hipEvent_t stop = nullptr;
  OR_SKIP_AND_RETURN(hipEventCreate(&start), NAME " failed to create event");
  OR_SKIP_AND_RETURN(hipEventCreate(&stop), NAME " failed to create event");

  // fixed number of loads regardless of stride
  read_t *src = nullptr;
  const size_t bytes = 1024ull * 1024ull * 1024ull * 2;
  const size_t size = bytes / sizeof(read_t); // number of read_t
  const int n = size / chunkSize;             // number of chunks in allocation
  const int dimBlock = 512;
  int dimGrid = (n + (dimBlock / 32) - 1) / (dimBlock / 32);
  dimGrid = min(dimGrid, 1024);


  // allocate on gpu0 and enable peer access
  OR_SKIP_AND_RETURN(hipSetDevice(gpu0), NAME "failed to set device");
  OR_SKIP_AND_RETURN(hipMalloc(&src, bytes),
                     NAME " failed to perform hipMalloc");
  OR_SKIP_AND_RETURN(hipMemset(src, 0, bytes),
                     NAME " failed to perform src hipMemset");
  if (gpu0 != gpu1) {
    hipError_t err = hipDeviceEnablePeerAccess(gpu1, 0);
    if (hipSuccess != err && hipErrorPeerAccessAlreadyEnabled != err) {
      state.SkipWithError(NAME " failed to ensure peer access");
    }
  }

  // enable peer access from gpu1
  OR_SKIP_AND_RETURN(hipSetDevice(gpu1), NAME "failed to set device");
  if (gpu0 != gpu1) {
    hipError_t err = hipDeviceEnablePeerAccess(gpu0, 0);
    if (hipSuccess != err && hipErrorPeerAccessAlreadyEnabled != err) {
      state.SkipWithError(NAME " failed to ensure peer access");
    }
  }

  // run pull kernel on dst device
  OR_SKIP_AND_RETURN(hipSetDevice(gpu1), NAME " unable to set pull device");

  for (auto _ : state) {
    // Start copy
    OR_SKIP_AND_BREAK(hipEventRecord(start, stream),
                      NAME " failed to record start event");

    Comm_chunk_pull_kernel<<<dimGrid, dimBlock, 0, stream>>>(
        src, chunkSize, chunkFill, n, nullptr);
    OR_SKIP_AND_BREAK(hipEventRecord(stop, stream),
                      NAME " failed to record stop event");

    // Wait for all copies to finish
    OR_SKIP_AND_BREAK(hipEventSynchronize(stop),
                      NAME " failed to synchronize");

    // Get the transfer time
    float millis;
    OR_SKIP_AND_BREAK(hipEventElapsedTime(&millis, start, stop),
                      NAME " failed to compute elapsed tiume");
    state.SetIterationTime(millis / 1000);
  }

  state.SetBytesProcessed(int64_t(state.iterations()) * n * chunkFill *
                          sizeof(read_t));
  state.counters["ld-bytes"] = n * chunkFill * sizeof(read_t);
  state.counters["ld-count"] = n * chunkFill;
  state.counters["ld-size"] = sizeof(read_t);
  state.counters["gpu0"] = gpu0;
  state.counters["gpu1"] = gpu1;
  state.counters["chunkSize"] = chunkSize;
  state.counters["chunkFill"] = chunkFill;
  state.counters["chunkCont"] = n;
  state.counters["dimgrid"] = dimGrid;

  OR_SKIP_AND_RETURN(hipEventDestroy(start), "hipEventDestroy");
  OR_SKIP_AND_RETURN(hipEventDestroy(stop), "hipEventDestroy");
  OR_SKIP_AND_RETURN(hipStreamDestroy(stream), "hipStreamDestroy");
  OR_SKIP_AND_RETURN(hipFree(src), "hipFree");

#if defined(SCOPE_USE_NVTX)
  nvtxRangePop();
#endif
};

static void registerer() {
  std::string name;
  const std::vector<MemorySpace> cudaSpaces = scope::system::memory_spaces(MemorySpace::Kind::cuda_device);

  for (const auto &space0 : cudaSpaces) {
    for (const auto &space1 : cudaSpaces) {

      auto gpu0 = space0.device_id();
      auto gpu1 = space1.device_id();
      int ok1, ok2;
      if (!PRINT_IF_ERROR(hipDeviceCanAccessPeer(&ok1, gpu0, gpu1)) &&
          !PRINT_IF_ERROR(hipDeviceCanAccessPeer(&ok2, gpu1, gpu0))) {
        if ((ok1 && ok2) || gpu0 == gpu1) {
          name = std::string(NAME) + "/" + std::to_string(gpu0) + "/" +
                 std::to_string(gpu1);
          benchmark::RegisterBenchmark(name.c_str(), Comm_chunk_pull, gpu0,
                                       gpu1)
              ->CHUNK_ARGS()
              ->UseManualTime();
        }
      }
    }
  }
}

SCOPE_AFTER_INIT(registerer, NAME);
