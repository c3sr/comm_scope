
#include <hip/hip_runtime.h>
#if __CUDACC_VER_MAJOR__ >= 8

#include "args.hpp"
#include "scope/scope.hpp"

#define NAME "Comm_UM_Demand_GPUToGPU"

template <bool NOOP = false>
__global__ void gpu_write(char *ptr, const size_t count, const size_t stride) {
  if (NOOP) {
    return;
  }

  // global ID
  const size_t gx = blockIdx.x * blockDim.x + threadIdx.x;
  // lane ID 0-31
  const size_t lx = gx & 31;
  // warp ID
  size_t wx = gx / 32;
  const size_t numWarps = (gridDim.x * blockDim.x + 32 - 1) / 32;

  if (0 == lx) {
    for (size_t i = wx * stride; i < count; i += numWarps * stride) {
      ptr[i] = 0;
    }
  }
}

auto Comm_UM_Demand_GPUToGPU = [](benchmark::State &state, const int src_gpu,
                                  const int dst_gpu) {
  const size_t pageSize = page_size();

  const auto bytes = 1ULL << static_cast<size_t>(state.range(0));

  if (PRINT_IF_ERROR(cuda_reset_device(src_gpu))) {
    state.SkipWithError(NAME " failed to reset CUDA src device");
    return;
  }
  if (PRINT_IF_ERROR(cuda_reset_device(dst_gpu))) {
    state.SkipWithError(NAME " failed to reset CUDA src device");
    return;
  }

  if (PRINT_IF_ERROR(cudaSetDevice(dst_gpu))) {
    state.SkipWithError(NAME " failed to set CUDA dst device");
    return;
  }

  char *ptr = nullptr;
  if (PRINT_IF_ERROR(cudaMallocManaged(&ptr, bytes))) {
    state.SkipWithError(NAME " failed to perform cudaMallocManaged");
    return;
  }
  defer(cudaFree(ptr));

  if (PRINT_IF_ERROR(cudaMemset(ptr, 0, bytes))) {
    state.SkipWithError(NAME " failed to perform cudaMemset");
    return;
  }

  cudaEvent_t start, stop;
  if (PRINT_IF_ERROR(cudaEventCreate(&start))) {
    state.SkipWithError(NAME " failed to create event");
    return;
  }
  defer(cudaEventDestroy(start));

  if (PRINT_IF_ERROR(cudaEventCreate(&stop))) {
    state.SkipWithError(NAME " failed to create event");
    return;
  }
  defer(cudaEventDestroy(stop));

  for (auto _ : state) {
    cudaMemPrefetchAsync(ptr, bytes, src_gpu);
    cudaSetDevice(src_gpu);
    cudaDeviceSynchronize();
    cudaSetDevice(dst_gpu);
    cudaDeviceSynchronize();
    if (PRINT_IF_ERROR(cudaGetLastError())) {
      state.SkipWithError(NAME " failed to prep iteration");
      return;
    }

    cudaEventRecord(start);
    gpu_write<<<256, 256>>>(ptr, bytes, pageSize);
    cudaEventRecord(stop);
    cudaEventSynchronize(stop);

    float millis = 0;
    if (PRINT_IF_ERROR(cudaEventElapsedTime(&millis, start, stop))) {
      state.SkipWithError(NAME " failed to get elapsed time");
      break;
    }
    state.SetIterationTime(millis / 1000);
  }

  state.SetBytesProcessed(int64_t(state.iterations()) * int64_t(bytes));
  state.counters["bytes"] = bytes;
  state.counters["src_gpu"] = src_gpu;
  state.counters["dst_gpu"] = dst_gpu;
};

static void registerer() {
  const std::vector<MemorySpace> cudaSpaces = scope::system::memory_spaces(MemorySpace::Kind::cuda_device);

  for (const auto &src : cudaSpaces) {
    for (const auto &dst : cudaSpaces) {

      auto src_gpu = src.device_id();
      auto dst_gpu = dst.device_id();
      std::string name = std::string(NAME) + "/" + std::to_string(src_gpu) +
                         "/" + std::to_string(dst_gpu);
      benchmark::RegisterBenchmark(name.c_str(), Comm_UM_Demand_GPUToGPU,
                                   src_gpu, dst_gpu)
          ->SMALL_ARGS()
          ->UseManualTime();
    }
  }
}

SCOPE_AFTER_INIT(registerer, NAME);

#endif // __CUDACC_VER_MAJOR__ >= 8
