
#include <hip/hip_runtime.h>
#if __CUDACC_VER_MAJOR__ >= 8

#include "scope/scope.hpp"

#include "args.hpp"

#define NAME "Comm_UM_Demand_HostToGPU"

template <bool NOOP = false>
__global__ void gpu_write(char *ptr, const size_t count, const size_t stride) {
  if (NOOP) {
    return;
  }

  // global ID
  const size_t gx = blockIdx.x * blockDim.x + threadIdx.x;
  // lane ID 0-31
  const size_t lx = gx & 31;
  // warp ID
  size_t wx = gx / 32;
  const size_t numWarps = (gridDim.x * blockDim.x + 32 - 1) / 32;

  if (0 == lx) {
    for (size_t i = wx * stride; i < count; i += numWarps * stride) {
      ptr[i] = 0;
    }
  }
}

auto Comm_UM_Demand_HostToGPU = [](benchmark::State &state, const int numa_id,
                                   const int cuda_id) {
  const size_t pageSize = page_size();

  const auto bytes = 1ULL << static_cast<size_t>(state.range(0));

  numa::ScopedBind binder(numa_id);

  if (PRINT_IF_ERROR(cuda_reset_device(cuda_id))) {
    state.SkipWithError(NAME " failed to reset device");
    return;
  }

  if (PRINT_IF_ERROR(cudaSetDevice(cuda_id))) {
    state.SkipWithError(NAME " failed to set CUDA dst device");
    return;
  }

  char *ptr = nullptr;
  if (PRINT_IF_ERROR(cudaMallocManaged(&ptr, bytes))) {
    state.SkipWithError(NAME " failed to perform cudaMallocManaged");
    return;
  }
  defer(cudaFree(ptr));

  if (PRINT_IF_ERROR(cudaMemset(ptr, 0, bytes))) {
    state.SkipWithError(NAME " failed to perform cudaMemset");
    return;
  }

  cudaEvent_t start, stop;
  if (PRINT_IF_ERROR(cudaEventCreate(&start))) {
    state.SkipWithError(NAME " failed to create start event");
    return;
  }
  defer(cudaEventDestroy(start));

  if (PRINT_IF_ERROR(cudaEventCreate(&stop))) {
    state.SkipWithError(NAME " failed to create end event");
    return;
  }
  defer(cudaEventDestroy(stop));

  for (auto _ : state) {
    cudaError_t err = cudaMemPrefetchAsync(ptr, bytes, cudaCpuDeviceId);
    if (err == cudaErrorInvalidDevice) {
      for (size_t i = 0; i < bytes; i += pageSize) {
        ptr[i] = 0;
      }
    }

    if (PRINT_IF_ERROR(cudaDeviceSynchronize())) {
      state.SkipWithError(NAME " failed to synchronize");
      return;
    }

    cudaEventRecord(start);
    gpu_write<<<256, 256>>>(ptr, bytes, pageSize);
    cudaEventRecord(stop);
    cudaEventSynchronize(stop);

    float millis = 0;
    if (PRINT_IF_ERROR(cudaEventElapsedTime(&millis, start, stop))) {
      state.SkipWithError(NAME " failed to get elapsed time");
      break;
    }
    state.SetIterationTime(millis / 1000);
  }

  state.SetBytesProcessed(int64_t(state.iterations()) * int64_t(bytes));
  state.counters["bytes"] = bytes;
  state.counters["cuda_id"] = cuda_id;
  state.counters["numa_id"] = numa_id;
};

static void registerer() {
  for (auto cuda_id : unique_cuda_device_ids()) {

    for (auto numa_id : numa::mems()) {

      std::string name = std::string(NAME) + "/" + std::to_string(numa_id) +
                         "/" + std::to_string(cuda_id);
      benchmark::RegisterBenchmark(name.c_str(), Comm_UM_Demand_HostToGPU,
                                   numa_id, cuda_id)
          ->SMALL_ARGS()
          ->UseManualTime();
    }
  }
}

SCOPE_AFTER_INIT(registerer, NAME);

#endif // __CUDACC_VER_MAJOR__ >= 8
