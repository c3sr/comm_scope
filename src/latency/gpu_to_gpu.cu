
#include <hip/hip_runtime.h>
#if __CUDACC_VER_MAJOR__ >= 8

#include <cassert>

#include <cuda_runtime.h>

#include "scope/scope.hpp"

#include "args.hpp"

#define NAME "Comm_UM_Latency_GPUToGPU"

template <bool NOOP = false>
__global__ void gpu_traverse(size_t *ptr, const size_t steps) {

  if (NOOP) {
    return;
  }
  size_t next = 0;
  for (int i = 0; i < steps; ++i) {
    next = ptr[next];
  }
  ptr[next] = 1;
}

auto Comm_UM_Latency_GPUToGPU = [](benchmark::State &state, const int src_gpu,
                                   const int dst_gpu) {
  if (src_gpu == dst_gpu) {
    state.SkipWithError(NAME "src and dst GPU should be different");
    return;
  }

  const size_t steps = state.range(0);
  const size_t stride = 65536 * 2;
  const size_t bytes = sizeof(size_t) * (steps + 1) * stride;

  if (PRINT_IF_ERROR(cuda_reset_device(src_gpu))) {
    state.SkipWithError(NAME " failed to reset src device");
    return;
  }
  if (PRINT_IF_ERROR(cuda_reset_device(src_gpu))) {
    state.SkipWithError(NAME " failed to reset dst device");
    return;
  }

  if (PRINT_IF_ERROR(cudaSetDevice(dst_gpu))) {
    state.SkipWithError(NAME " failed to set CUDA dst device");
    return;
  }

  size_t *ptr = nullptr;
  if (PRINT_IF_ERROR(cudaMallocManaged(&ptr, bytes))) {
    state.SkipWithError(NAME " failed to perform cudaMallocManaged");
    return;
  }
  defer(cudaFree(ptr));

  if (PRINT_IF_ERROR(cudaMemset(ptr, 0, bytes))) {
    state.SkipWithError(NAME " failed to perform cudaMemset");
    return;
  }
  // set up stride pattern
  for (size_t i = 0; i < steps; ++i) {
    ptr[i * stride] = (i + 1) * stride;
  }
  if (PRINT_IF_ERROR(cudaSetDevice(src_gpu))) {
    state.SkipWithError(NAME " failed to set CUDA src device");
    return;
  }
  if (PRINT_IF_ERROR(cudaDeviceSynchronize())) {
    state.SkipWithError(NAME " failed to synchronize");
    return;
  }

  if (PRINT_IF_ERROR(cudaSetDevice(dst_gpu))) {
    state.SkipWithError(NAME " failed to set CUDA dst device");
    return;
  }
  cudaEvent_t start, stop;
  if (PRINT_IF_ERROR(cudaEventCreate(&start))) {
    state.SkipWithError(NAME " failed to create start event");
    return;
  }
  defer(cudaEventDestroy(start));

  if (PRINT_IF_ERROR(cudaEventCreate(&stop))) {
    state.SkipWithError(NAME " failed to create end event");
    return;
  }
  defer(cudaEventDestroy(stop));

  for (auto _ : state) {
    if (PRINT_IF_ERROR(cudaMemPrefetchAsync(ptr, bytes, src_gpu))) {
      state.SkipWithError(NAME " failed to prefetch to src");
      return;
    }
    if (PRINT_IF_ERROR(cudaSetDevice(src_gpu))) {
      state.SkipWithError(NAME " failed to set CUDA src device");
      return;
    }
    if (PRINT_IF_ERROR(cudaDeviceSynchronize())) {
      state.SkipWithError(NAME " failed to synchronize");
      return;
    }
    if (PRINT_IF_ERROR(cudaSetDevice(dst_gpu))) {
      state.SkipWithError(NAME " failed to set dst device");
      return;
    }
    if (PRINT_IF_ERROR(cudaDeviceSynchronize())) {
      state.SkipWithError(NAME " failed to synchronize");
      return;
    }

    cudaEventRecord(start);
    gpu_traverse<<<1, 1>>>(ptr, steps);
    cudaEventRecord(stop);
    cudaEventSynchronize(stop);

    float millis = 0;
    if (PRINT_IF_ERROR(cudaEventElapsedTime(&millis, start, stop))) {
      state.SkipWithError(NAME " failed to get elapsed time");
      break;
    }
    state.SetIterationTime(millis / 1000);
    state.counters["strides"] = steps;
    state.counters["src_gpu"] = src_gpu;
    state.counters["dst_gpu"] = dst_gpu;
  }
};

static void registerer() {
  const std::vector<Device> cudas = scope::system::cuda_devices();
  for (size_t i = 0; i < cudas.size(); ++i) {
    for (size_t j = i + 1; j < cudas.size(); ++j) {
      auto src_gpu = cudas[i];
      auto dst_gpu = cudas[j];
      std::string name = std::string(NAME) + "/" + std::to_string(src_gpu) +
                         "/" + std::to_string(dst_gpu);
      benchmark::RegisterBenchmark(name.c_str(), Comm_UM_Latency_GPUToGPU,
                                   src_gpu, dst_gpu)
          ->SMALL_ARGS()
          ->UseManualTime();
    }
  }
}

SCOPE_AFTER_INIT(registerer, NAME);

#endif // __CUDACC_VER_MAJOR__ >= 8