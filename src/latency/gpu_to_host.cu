
#include <hip/hip_runtime.h>
#if __CUDACC_VER_MAJOR__ >= 8

 #include "scope/scope.hpp"

#include "args.hpp"

#define NAME "Comm_UM_Latency_GPUToHost"

template <bool NOOP = false>
void cpu_traverse(size_t *ptr, const size_t steps) {

  if (NOOP) {
    return;
  }
  size_t next = 0;
  for (size_t i = 0; i < steps; ++i) {
    next = ptr[next];
  }
  ptr[next] = 1;
}

auto Comm_UM_Latency_GPUToHost = [] (benchmark::State &state,
  const int numa_id,
  const int cuda_id) {

  const size_t steps = state.range(0);

  const size_t stride = 65536 * 2;
  const size_t bytes  = sizeof(size_t) * (steps + 1) * stride;
  numa::ScopedBind binder(numa_id);
  if (PRINT_IF_ERROR(cudaSetDevice(cuda_id))) {
    state.SkipWithError(NAME " failed to set CUDA device");
    return;
  }
  if (PRINT_IF_ERROR(cudaDeviceReset())) {
    state.SkipWithError(NAME " failed to reset device");
    return;
  }

  size_t *ptr = nullptr;
  if (PRINT_IF_ERROR(cudaMallocManaged(&ptr, bytes))) {
    state.SkipWithError(NAME " failed to perform cudaMallocManaged");
    return;
  }
  defer(cudaFree(ptr));

  if (PRINT_IF_ERROR(cudaMemset(ptr, 0, bytes))) {
    state.SkipWithError(NAME " failed to perform cudaMemset");
    return;
  }

  // set up stride pattern
  for (size_t i = 0; i < steps; ++i) {
    ptr[i * stride] = (i + 1) * stride;
  }
  if (PRINT_IF_ERROR(cudaDeviceSynchronize())) {
    state.SkipWithError(NAME " failed to synchronize");
    return;
  }

  for (auto _ : state) {
    state.PauseTiming();
    // prefetch to source
    if (PRINT_IF_ERROR(cudaMemPrefetchAsync(ptr, bytes, cuda_id))) {
      state.SkipWithError(NAME " failed prefetch to device");
      return;
    }
    if (PRINT_IF_ERROR(cudaDeviceSynchronize())) {
      state.SkipWithError(NAME " failed to synchronize");
      return;
    }
    state.ResumeTiming();

    cpu_traverse(ptr, steps);
  }
  state.counters["strides"] = steps;
  state.counters["cuda_id"] = cuda_id;
  state.counters["numa_id"] = numa_id;
};

static void registerer() {
  for (int cuda_id : scope::system::cuda_devices()) {
    for (auto numa_id : numa::mems()) {
      std::string name = std::string(NAME)
                       + "/" + std::to_string(numa_id) 
                       + "/" + std::to_string(cuda_id);
      benchmark::RegisterBenchmark(name.c_str(), Comm_UM_Latency_GPUToHost,
        numa_id,
        cuda_id)->SMALL_ARGS();
    }
  }
}

SCOPE_AFTER_INIT(registerer, NAME);

#endif // __CUDACC_VER_MAJOR__ >= 8