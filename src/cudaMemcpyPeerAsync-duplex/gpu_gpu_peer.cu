#include "hip/hip_runtime.h"
#include <cassert>
#include <hip/hip_runtime.h>

#include "scope/scope.hpp"

#include "args.hpp"

#define NAME "Comm_cudaMemcpyPeerAsync_Duplex_GPUGPUPeer"

__global__ void busy_wait(clock_t *d, clock_t clock_count) {
  clock_t start_clock = clock64();
  clock_t clock_offset = 0;
  while (clock_offset < clock_count) {
    clock_offset = clock64() - start_clock;
  }
  if (d) {
    *d = clock_offset;
  }
}

auto Comm_cudaMemcpyPeerAsync_Duplex_GPUGPUPeer = [](benchmark::State &state,
                                                     const int gpu0,
                                                     const int gpu1) {
  const auto bytes = 1ULL << static_cast<size_t>(state.range(0));

  OR_SKIP_AND_RETURN(cuda_reset_device(gpu0),
                     "failed to reset src CUDA device");
  OR_SKIP_AND_RETURN(cuda_reset_device(gpu1),
                     "failed to reset dst CUDA device");

  void *src0 = nullptr;
  void *src1 = nullptr;
  void *dst0 = nullptr;
  void *dst1 = nullptr;
  hipStream_t stream0;
  hipStream_t stream1;
  hipError_t err;
  hipEvent_t start, stop1, stop;

  OR_SKIP_AND_RETURN(hipSetDevice(gpu0), "failed to set src device");
  OR_SKIP_AND_RETURN(hipMalloc(&src0, bytes),
                     "failed to perform src0 hipMalloc");
  OR_SKIP_AND_RETURN(hipMalloc(&dst0, bytes),
                     "failed to perform src1 hipMalloc");
  defer(hipFree(src0));
  defer(hipFree(dst0));
  OR_SKIP_AND_RETURN(hipMemset(src0, 0, bytes),
                     "failed to perform src0 hipMemset");
  OR_SKIP_AND_RETURN(hipMemset(dst0, 0, bytes),
                     "failed to perform src1 hipMemset");
  OR_SKIP_AND_RETURN(hipStreamCreate(&stream0), "failed to create stream");
  defer(hipStreamDestroy(stream0));
  OR_SKIP_AND_RETURN(hipEventCreate(&start), "couldn't create start event");
  OR_SKIP_AND_RETURN(hipEventCreate(&stop), "couldn't create stop event");
  defer(hipEventDestroy(start));
  defer(hipEventDestroy(stop));
  if (gpu0 != gpu1) {
    err = hipDeviceEnablePeerAccess(gpu1, 0);
    hipGetLastError(); // clear error
    if (hipSuccess != err && hipErrorPeerAccessAlreadyEnabled != err) {
      state.SkipWithError("failed to enable peer access to gpu1");
      return;
    }
  }

  OR_SKIP_AND_RETURN(hipSetDevice(gpu1), "failed to set dst device");
  OR_SKIP_AND_RETURN(hipMalloc(&src1, bytes),
                     "failed to perform src1 hipMalloc");
  OR_SKIP_AND_RETURN(hipMalloc(&dst1, bytes),
                     "failed to perform dst1 hipMalloc");
  defer(hipFree(src1));
  defer(hipFree(dst1));
  OR_SKIP_AND_RETURN(hipMemset(src1, 0, bytes),
                     "failed to perform dst hipMemset");
  OR_SKIP_AND_RETURN(hipMemset(dst1, 0, bytes),
                     "failed to perform dst hipMemset");
  OR_SKIP_AND_RETURN(hipStreamCreate(&stream1), "failed to create stream");
  defer(hipStreamDestroy(stream1));
  OR_SKIP_AND_RETURN(hipEventCreate(&stop1), "couldn't create stop1 event");
  defer(hipEventDestroy(stop1));
  if (gpu0 != gpu1) {
    err = hipDeviceEnablePeerAccess(gpu0, 0);
    hipGetLastError(); // clear error
    if (hipSuccess != err && hipErrorPeerAccessAlreadyEnabled != err) {
      state.SkipWithError("failed to enable peer access to gpu0");
      return;
    }
  }

  size_t cycles = 4096;
  for (auto _ : state) {

    // keep making kernel longer and longer until it hides all host code
    restart_iteration:
      OR_SKIP_AND_BREAK(hipSetDevice(gpu0), "failed to set src device");
      busy_wait<<<1, 1, 0, stream0>>>(nullptr, cycles);
      OR_SKIP_AND_BREAK(hipGetLastError(), "failed to busy_wait");
      OR_SKIP_AND_BREAK(hipEventRecord(start, stream0),
                        "failed to record start");
      OR_SKIP_AND_BREAK(
          hipMemcpyPeerAsync(dst1, gpu1, src0, gpu0, bytes, stream0),
          "failed to memcpy");
      OR_SKIP_AND_BREAK(hipSetDevice(gpu1), "failed to set src device");
      OR_SKIP_AND_BREAK(hipStreamWaitEvent(stream1, start, 0),
                        "failed to wait");
      OR_SKIP_AND_BREAK(
          hipMemcpyPeerAsync(dst0, gpu0, src1, gpu1, bytes, stream1),
          "failed to memcpy");
      OR_SKIP_AND_BREAK(hipEventRecord(stop1, stream1), "failed to stop");
      OR_SKIP_AND_BREAK(hipSetDevice(gpu0), "failed to set src device");
      OR_SKIP_AND_BREAK(hipStreamWaitEvent(stream0, stop1, 0),
                        "failed to set src device");
      OR_SKIP_AND_BREAK(hipEventRecord(stop, stream0), "failed to stop");

      // if kernel has ended, it wasn't long enough to cover the host code:
      // finish transfers, increase cycles, and try again
      err = hipEventQuery(start);
      if (hipSuccess == err) {
        cycles *= 2;
        OR_SKIP_AND_BREAK(hipStreamSynchronize(stream0),
                          "failed to wait for stream0");
        OR_SKIP_AND_BREAK(hipStreamSynchronize(stream1),
                          "failed to wait for stream1");
        goto restart_iteration;
      } else if (hipErrorNotReady == err) {
        // kernel was long enough
      } else {
        OR_SKIP_AND_BREAK(err, "errored while waiting for kernel");
      }

    OR_SKIP_AND_BREAK(hipEventSynchronize(stop), "failed to synchronize");
    float ms = 0.0f;
    OR_SKIP_AND_BREAK(hipEventElapsedTime(&ms, start, stop),
                      NAME "failed to compute elapsed time");
    state.SetIterationTime(ms / 1000);
  }
  state.SetBytesProcessed(int64_t(state.iterations()) * int64_t(bytes) * 2);
  state.counters["bytes"] = bytes;
  state.counters["gpu0"] = gpu0;
  state.counters["gpu1"] = gpu1;
  state.counters["wait_cycles"] = cycles;
};

static void registerer() {
  std::string name;
  const std::vector<MemorySpace> cudaSpaces = scope::system::memory_spaces(MemorySpace::Kind::cuda_device);

  for (const auto &space0 : cudaSpaces) {
    for (const auto &space1 : cudaSpaces) {

      auto gpu0 = space0.device_id();
      auto gpu1 = space1.device_id();
      name = std::string(NAME) + "/" + std::to_string(gpu0) + "/" +
             std::to_string(gpu1);
      benchmark::RegisterBenchmark(
          name.c_str(), Comm_cudaMemcpyPeerAsync_Duplex_GPUGPUPeer, gpu0, gpu1)
          ->SMALL_ARGS()
          ->UseManualTime();
    }
  }
}

SCOPE_AFTER_INIT(registerer, NAME);
