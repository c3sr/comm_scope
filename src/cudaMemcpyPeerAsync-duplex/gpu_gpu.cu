#include "hip/hip_runtime.h"
/* it seems that above certain sizes, hipMemcpyPeerAsync is not async 
  so limit to 2^27 
*/
#include <hip/hip_runtime.h>

#include "scope/scope.hpp"

#include "args.hpp"

#define NAME "Comm_cudaMemcpyPeerAsync_Duplex_GPUGPU"

namespace comm_cudaMemcpyPeerAsync_Duplex_GPUGPU {
__global__ void busy_wait(clock_t *d, clock_t clock_count) {
  clock_t start_clock = clock64();
  clock_t clock_offset = 0;
  while (clock_offset < clock_count) {
    clock_offset = clock64() - start_clock;
  }
  if (d) {
    *d = clock_offset;
  }
}
} // namespace comm_cudaMemcpyPeerAsync_Duplex_GPUGPU


auto Comm_cudaMemcpyPeerAsync_Duplex_GPUGPU = [](benchmark::State &state,
                                                 const int gpu0,
                                                 const int gpu1) {
  const auto bytes = 1ULL << static_cast<size_t>(state.range(0));

  OR_SKIP_AND_RETURN(scope::cuda_reset_device(gpu0),
                     NAME " failed to reset src CUDA device");
  OR_SKIP_AND_RETURN(scope::cuda_reset_device(gpu1),
                     NAME " failed to reset dst CUDA device");

  void *src0 = nullptr;
  void *src1 = nullptr;
  void *dst0 = nullptr;
  void *dst1 = nullptr;
  hipStream_t stream0;
  hipStream_t stream1;
  hipError_t err;
  hipEvent_t start, stop1, stop;

  OR_SKIP_AND_RETURN(hipSetDevice(gpu0), NAME " failed to set src device");
  OR_SKIP_AND_RETURN(hipMalloc(&src0, bytes),
                     NAME " failed to perform src0 hipMalloc");
  OR_SKIP_AND_RETURN(hipMalloc(&dst0, bytes),
                     NAME " failed to perform src1 hipMalloc");
  defer(hipFree(src0));
  defer(hipFree(dst0));
  OR_SKIP_AND_RETURN(hipMemset(src0, 0, bytes),
                     NAME " failed to perform src0 hipMemset");
  OR_SKIP_AND_RETURN(hipMemset(dst0, 0, bytes),
                     NAME " failed to perform src1 hipMemset");
  OR_SKIP_AND_RETURN(hipStreamCreate(&stream0),
                     NAME " failed to create stream");
  defer(hipStreamDestroy(stream0));
  OR_SKIP_AND_RETURN(hipEventCreate(&start),
                     NAME " couldn't create start event");
  OR_SKIP_AND_RETURN(hipEventCreate(&stop),
                     NAME " couldn't create stop event");
  defer(hipEventDestroy(start));
  defer(hipEventDestroy(stop));
  if (gpu0 != gpu1) {
    err = hipDeviceDisablePeerAccess(gpu1);
    hipGetLastError(); // clear error
    if (hipSuccess != err && hipErrorPeerAccessNotEnabled != err) {
      state.SkipWithError(NAME " failed to disable peer access");
      return;
    }
  }

  OR_SKIP_AND_RETURN(hipSetDevice(gpu1), NAME " failed to set dst device");
  OR_SKIP_AND_RETURN(hipMalloc(&src1, bytes),
                     NAME " failed to perform src1 hipMalloc");
  OR_SKIP_AND_RETURN(hipMalloc(&dst1, bytes),
                     NAME " failed to perform dst1 hipMalloc");
  defer(hipFree(src1));
  defer(hipFree(dst1));
  OR_SKIP_AND_RETURN(hipMemset(src1, 0, bytes),
                     NAME " failed to perform dst hipMemset");
  OR_SKIP_AND_RETURN(hipMemset(dst1, 0, bytes),
                     NAME " failed to perform dst hipMemset");
  OR_SKIP_AND_RETURN(hipStreamCreate(&stream1),
                     NAME " failed to create stream");
  defer(hipStreamDestroy(stream1));
  OR_SKIP_AND_RETURN(hipEventCreate(&stop1),
                     NAME " couldn't create stop1 event");
  defer(hipEventDestroy(stop1));
  if (gpu0 != gpu1) {
    err = hipDeviceDisablePeerAccess(gpu0);
    hipGetLastError(); // clear error
    if (hipSuccess != err && hipErrorPeerAccessNotEnabled != err) {
      state.SkipWithError(NAME " failed to disable peer access");
      return;
    }
  }

  size_t cycles = 4096;
  for (auto _ : state) {
    // make kernel longer until it hides host code
    restart_iteration: // use a label here instead of a nested loop so OR_SKIP_AND_BREAK breaks out of benchmark loop
      OR_SKIP_AND_BREAK(hipSetDevice(gpu0), NAME " failed to set src device");
      comm_cudaMemcpyPeerAsync_Duplex_GPUGPU::busy_wait<<<1, 1, 0, stream0>>>(
          nullptr, cycles);
      OR_SKIP_AND_BREAK(hipGetLastError(), NAME " failed to busy_wait");
      OR_SKIP_AND_BREAK(hipEventRecord(start, stream0),
                        NAME " failed to record start");
      OR_SKIP_AND_BREAK(
          hipMemcpyPeerAsync(dst1, gpu1, src0, gpu0, bytes, stream0),
          NAME " failed to memcpy");
      OR_SKIP_AND_BREAK(hipSetDevice(gpu1), NAME " failed to set src device");
      OR_SKIP_AND_BREAK(hipStreamWaitEvent(stream1, start, 0),
                        NAME " failed to set src device");
      OR_SKIP_AND_BREAK(
          hipMemcpyPeerAsync(dst0, gpu0, src1, gpu1, bytes, stream1),
          NAME " failed to memcpy");
      OR_SKIP_AND_BREAK(hipEventRecord(stop1, stream1),
                        NAME " failed to stop");
      OR_SKIP_AND_BREAK(hipSetDevice(gpu0), NAME " failed to set src device");
      OR_SKIP_AND_BREAK(hipStreamWaitEvent(stream0, stop1, 0),
                        NAME " failed to set src device");
      OR_SKIP_AND_BREAK(hipEventRecord(stop, stream0), NAME " failed to stop");

      // if kernel has ended, it wasn't long enough to cover the host code.
      // finish transfers, increase cycles, and try again
      err = hipEventQuery(start);
      if (hipSuccess == err) {
        cycles *= 2;
        OR_SKIP_AND_BREAK(hipStreamSynchronize(stream0),
                          NAME " failed to wait for stream0");
        OR_SKIP_AND_BREAK(hipStreamSynchronize(stream1),
                          NAME " failed to wait for stream1");
        goto restart_iteration;
      } else if (hipErrorNotReady == err) {
        // kernel was long enough
      } else {
        OR_SKIP_AND_BREAK(err, NAME " errored while waiting for kernel");
      }

    OR_SKIP_AND_BREAK(hipEventSynchronize(stop),
                      NAME " failed to synchronize");

    float ms = 0.0f;
    OR_SKIP_AND_BREAK(hipEventElapsedTime(&ms, start, stop),
                      NAME "failed to compute elapsed time");
    state.SetIterationTime(ms / 1000);
  }
  state.SetBytesProcessed(int64_t(state.iterations()) * int64_t(bytes) * 2);
  state.counters["bytes"] = bytes;
  state.counters["gpu0"] = gpu0;
  state.counters["gpu1"] = gpu1;
  state.counters["wait_cycles"] = cycles;
};


static void registerer() {
  const std::vector<MemorySpace> cudaSpaces = scope::system::memory_spaces(MemorySpace::Kind::cuda_device);

  for (const auto &space0 : cudaSpaces) {
    for (const auto &space1 : cudaSpaces) {

      auto gpu0 = space0.device_id();
      auto gpu1 = space1.device_id();
      std::string name = std::string(NAME) + "/" + std::to_string(gpu0) + "/" +
      std::to_string(gpu1);
      benchmark::RegisterBenchmark(
        name.c_str(), Comm_cudaMemcpyPeerAsync_Duplex_GPUGPU, gpu0, gpu1)
        ->SMALL_ARGS()
        ->UseManualTime();
      }
    }
  }

SCOPE_AFTER_INIT(registerer, NAME);
