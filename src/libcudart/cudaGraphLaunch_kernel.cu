#include "hip/hip_runtime.h"


#include "scope/scope.hpp"

#include "args.hpp"

#define NAME "Comm_cudart_cudaGraphLaunch_kernel"

__global__ void Comm_cudart_cudaGraphLaunch_kernel_kernel() {}

auto Comm_cudart_cudaGraphLaunch_kernel = [](benchmark::State &state,
                                             const int numaId,
                                             const int cudaId) {
  const int iters = state.range(0);

  numa::ScopedBind binder(numaId);

  OR_SKIP_AND_RETURN(cuda_reset_device(cudaId), "failed to reset device");
  OR_SKIP_AND_RETURN(hipSetDevice(cudaId), "failed to set CUDA dst device");

  hipGraph_t graph;
  hipGraphExec_t instance;
  hipStream_t stream;
  OR_SKIP_AND_RETURN(hipStreamCreate(&stream), "failed to create stream");

  // create the graph to launch
  OR_SKIP_AND_RETURN(hipStreamBeginCapture(stream
#if __CUDACC_VER_MAJOR__ >= 11 ||                                              \
    (__CUDACC_VER_MAJOR__ >= 10 && __CUDACC_VER_MINOR__ > 0)
                                            ,
                                            hipStreamCaptureModeGlobal
#endif
                                            ),
                     "");
  for (int i = 0; i < iters; ++i) {
    Comm_cudart_cudaGraphLaunch_kernel_kernel<<<1, 1, 0, stream>>>();
  }
  OR_SKIP_AND_RETURN(hipStreamEndCapture(stream, &graph), "");
  OR_SKIP_AND_RETURN(hipGraphInstantiate(&instance, graph, NULL, NULL, 0), "");
  defer(hipGraphDestroy(graph));
  defer(hipGraphExecDestroy(instance));

  for (auto _ : state) {
    state.PauseTiming();
    OR_SKIP_AND_BREAK(hipStreamSynchronize(stream), "");
    state.ResumeTiming();
    OR_SKIP_AND_BREAK(hipGraphLaunch(instance, stream), "");
  }

  state.SetItemsProcessed(state.iterations());
  state.counters["cuda_id"] = cudaId;
  state.counters["numa_id"] = numaId;
};

static void registerer() {
  for (int cudaId : scope::system::cuda_devices()) {
    for (auto numaId : numa::mems()) {
      std::string name = std::string(NAME) + "/" + std::to_string(numaId) +
                         "/" + std::to_string(cudaId);
      benchmark::RegisterBenchmark(
          name.c_str(), Comm_cudart_cudaGraphLaunch_kernel, numaId, cudaId)
          ->GRAPH_ARGS();
    }
  }
}

SCOPE_AFTER_INIT(registerer, NAME);
