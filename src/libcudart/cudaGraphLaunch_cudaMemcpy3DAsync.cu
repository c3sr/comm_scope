/* compiled with nvcc so __CUDACC_VER_MAJOR__ is defined
 */

#include "scope/scope.hpp"

#include "args.hpp"

#include <hip/hip_runtime.h>

#define NAME "Comm_cudart_cudaGraphLaunch_cudaMemcpy3DAsync"

auto Comm_cudart_cudaGraphLaunch_cudaMemcpy3DAsync = [](benchmark::State &state,
                                                        const int gpu0,
                                                        const int gpu1) {
  const int iters = state.range(0);

  OR_SKIP_AND_RETURN(cuda_reset_device(gpu0),
                     NAME " failed to reset CUDA device");
  OR_SKIP_AND_RETURN(cuda_reset_device(gpu1),
                     NAME " failed to reset CUDA device");

  hipGetLastError(); // clear any previous errors
  OR_SKIP_AND_RETURN(hipGetLastError(), "last error");

  // small enough transfer that the runtime cost is larger
  hipExtent copyExt;
  copyExt.width = 8;
  copyExt.height = 8;
  copyExt.depth = 8;

  // properties of the allocation
  hipExtent allocExt;
  allocExt.width = copyExt.width;
  allocExt.height = copyExt.height;
  allocExt.depth = copyExt.depth;

  hipMemcpy3DParms params = {};

  // allocate on gpu0 and enable peer access
  OR_SKIP_AND_RETURN(hipSetDevice(gpu0), NAME "failed to set device");
  OR_SKIP_AND_RETURN(hipMalloc3D(&params.srcPtr, allocExt),
                     NAME " failed to perform hipMalloc3D");
  allocExt.width = params.srcPtr.pitch;
  OR_SKIP_AND_RETURN(hipMemset3D(params.srcPtr, 0, allocExt),
                     NAME " failed to perform src hipMemset");
  if (gpu0 != gpu1) {
    hipError_t err = hipDeviceEnablePeerAccess(gpu1, 0);
    if (hipSuccess != err && hipErrorPeerAccessAlreadyEnabled != err) {
      state.SkipWithError(NAME " failed to ensure peer access");
    }
  }

  hipStream_t stream;
  OR_SKIP_AND_RETURN(hipStreamCreate(&stream), NAME "failed to create stream");
  defer(hipStreamDestroy(stream));

  // allocate on gpu1 and enable peer access
  OR_SKIP_AND_RETURN(hipSetDevice(gpu1), NAME "failed to set device");
  OR_SKIP_AND_RETURN(hipMalloc3D(&params.dstPtr, allocExt),
                     NAME " failed to perform hipMalloc3D");
  OR_SKIP_AND_RETURN(hipMemset3D(params.dstPtr, 0, allocExt),
                     NAME " failed to perform src hipMemset");
  if (gpu0 != gpu1) {
    hipError_t err = hipDeviceEnablePeerAccess(gpu0, 0);
    if (hipSuccess != err && hipErrorPeerAccessAlreadyEnabled != err) {
      state.SkipWithError(NAME " failed to ensure peer access");
    }
  }

  // set up copy parameters
  params.dstArray = 0; // provided dstPtr
  params.srcArray = 0; // provided srcPtr
  params.dstPos = make_hipPos(0, 0, 0);
  params.srcPos = make_hipPos(0, 0, 0);
  params.extent = copyExt;
  params.kind = hipMemcpyDeviceToDevice;

  hipGraph_t graph = 0;
  hipGraphExec_t instance = 0;

  // create the graph to launch
  OR_SKIP_AND_RETURN(hipSetDevice(gpu0), "failed to set device");
  OR_SKIP_AND_RETURN(hipStreamBeginCapture(stream
#if __CUDACC_VER_MAJOR__ >= 11 ||                                              \
    (__CUDACC_VER_MAJOR__ >= 10 && __CUDACC_VER_MINOR__ > 0)
                                            ,
                                            hipStreamCaptureModeThreadLocal
#endif
                                            ),
                     "");

  hipError_t err = hipSuccess; // if iters is 0
  for (int i = 0; i < iters; ++i) {
    err = hipMemcpy3DAsync(&params, stream);
  }
  OR_SKIP_AND_RETURN(err, "?"); // FIXME: what is the purpose of this little looper

  OR_SKIP_AND_RETURN(hipStreamEndCapture(stream, &graph), "capture error");
  OR_SKIP_AND_RETURN(hipGraphInstantiate(&instance, graph, NULL, NULL, 0),
                     "error during instantiate");
  defer(hipGraphDestroy(graph));
  defer(hipGraphExecDestroy(instance));

  for (auto _ : state) {
    state.PauseTiming();
    OR_SKIP_AND_BREAK(hipStreamSynchronize(stream), "error in sync");
    state.ResumeTiming();
    OR_SKIP_AND_BREAK(hipGraphLaunch(instance, stream), "error in launch");
  }

  state.SetItemsProcessed(state.iterations());
  state.counters["gpu0"] = gpu0;
  state.counters["gpu1"] = gpu1;
};

static void registerer() {
  std::string name;

  const std::vector<Device> cudas = scope::system::cuda_devices();

  for (size_t i = 0; i < cudas.size(); ++i) {
    for (size_t j = i; j < cudas.size(); ++j) {
      int gpu0 = cudas[i];
      int gpu1 = cudas[j];
      int ok1, ok2;
      if (!PRINT_IF_ERROR(hipDeviceCanAccessPeer(&ok1, gpu0, gpu1)) &&
          !PRINT_IF_ERROR(hipDeviceCanAccessPeer(&ok2, gpu1, gpu0))) {
        if ((ok1 && ok2) || i == j) {
          name = std::string(NAME) + "/" + std::to_string(gpu0) + "/" +
                 std::to_string(gpu1);
          benchmark::RegisterBenchmark(
              name.c_str(), Comm_cudart_cudaGraphLaunch_cudaMemcpy3DAsync, gpu0,
              gpu1)
              ->UseRealTime()
              ->GRAPH_ARGS();
        }
      }
    }
  }
}

SCOPE_AFTER_INIT(registerer, NAME);
