/* compiled with nvcc for __CUDACC_VER_MAJOR__
 */
#include "scope/scope.hpp"

#include "args.hpp"

#define NAME "Comm_cudart_cudaGraphInstantiate_cudaMemcpyAsync"

auto Comm_cudart_cudaGraphInstantiate_cudaMemcpyAsync =
    [](benchmark::State &state, const int numaId, const int cudaId) {
      const int iters = state.range(0);

      numa::ScopedBind binder(numaId);

      OR_SKIP_AND_RETURN(scope::cuda_reset_device(cudaId), "failed to reset device");
      OR_SKIP_AND_RETURN(hipSetDevice(cudaId),
                         "failed to set CUDA dst device");

      hipGraph_t graph;
      hipGraphExec_t instance;
      hipStream_t stream;
      OR_SKIP_AND_RETURN(hipStreamCreate(&stream), "failed to create stream");

      void *src = nullptr;
      void *dst = nullptr;

      OR_SKIP_AND_RETURN(hipMalloc(&src, 100), "");
      OR_SKIP_AND_RETURN(hipMalloc(&dst, 100), "");
      defer(hipFree(src));
      defer(hipFree(dst));

      hipError_t err;
      for (auto _ : state) {
        state.PauseTiming();
        OR_SKIP_AND_BREAK(hipStreamBeginCapture(stream
#if __CUDACC_VER_MAJOR__ >= 11 ||                                              \
    (__CUDACC_VER_MAJOR__ >= 10 && __CUDACC_VER_MINOR__ > 0)
                                                 ,
                                                 hipStreamCaptureModeGlobal
#endif
                                                 ),
                          "");
        for (int i = 0; i < iters; ++i) {
          hipMemcpyAsync(dst, src, 100, hipMemcpyDefault, stream);
        }
        OR_SKIP_AND_BREAK(hipStreamEndCapture(stream, &graph), "");
        state.ResumeTiming();
        err = hipGraphInstantiate(&instance, graph, NULL, NULL, 0);
        state.PauseTiming();
        OR_SKIP_AND_BREAK(hipGraphDestroy(graph), "");
        OR_SKIP_AND_BREAK(hipGraphExecDestroy(instance), "");
        state.ResumeTiming();
      }
      OR_SKIP_AND_RETURN(err, "failed to hipGraphInstantiate");

      state.SetItemsProcessed(state.iterations());
      state.counters["cuda_id"] = cudaId;
      state.counters["numa_id"] = numaId;
    };

static void registerer() {
  for (int cudaId : scope::system::cuda_devices()) {
    for (auto numaId : numa::mems()) {
      std::string name = std::string(NAME) + "/" + std::to_string(numaId) +
                         "/" + std::to_string(cudaId);
      benchmark::RegisterBenchmark(
          name.c_str(), Comm_cudart_cudaGraphInstantiate_cudaMemcpyAsync,
          numaId, cudaId)
          ->GRAPH_ARGS();
    }
  }
}

SCOPE_AFTER_INIT(registerer, NAME);
