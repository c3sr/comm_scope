#include "hip/hip_runtime.h"
#include <sstream>

#include "scope/scope.hpp"

#include "args.hpp"

#define NAME "Comm_stride_pull"

typedef int read_t;

static __global__ void Comm_stride_pull_kernel(read_t *__restrict__ src,
                                               const int n, // number of reads
                                               const int stride, read_t *flag) {

  for (int i = blockDim.x * blockIdx.x + threadIdx.x; i < n;
       i += gridDim.x * blockDim.x) {
    read_t t;
    do_not_optimize(t = src[i * stride]);
    if (flag) {
      *flag = t;
    }
  }
}

auto Comm_stride_pull = [](benchmark::State &state, const int gpu0,
                           const int gpu1) {
  const int stride = state.range(0);

#if defined(SCOPE_USE_NVTX)
  {
    std::stringstream name;
    name << NAME << "/" << gpu0 << "/" << gpu1 << "/" << state.range(0) << "/"
         << state.range(1) << "/" << state.range(2);
    nvtxRangePush(name.str().c_str());
  }
#endif

  OR_SKIP_AND_RETURN(scope::cuda_reset_device(gpu0),
                     NAME " failed to reset CUDA device");
  OR_SKIP_AND_RETURN(scope::cuda_reset_device(gpu1),
                     NAME " failed to reset CUDA device");

  // create stream on dst gpu (pull)
  OR_SKIP_AND_RETURN(hipSetDevice(gpu1), NAME "failed to create stream");
  hipStream_t stream = nullptr;
  OR_SKIP_AND_RETURN(hipStreamCreate(&stream), NAME "failed to create stream");

  // Start and stop events on dst gpu (pull)
  hipEvent_t start = nullptr;
  hipEvent_t stop = nullptr;
  OR_SKIP_AND_RETURN(hipEventCreate(&start), NAME " failed to create event");
  OR_SKIP_AND_RETURN(hipEventCreate(&stop), NAME " failed to create event");

  // fixed number of loads regardless of stride
  read_t *src = nullptr;
  const size_t bytes = 1024ull * 1024ull * 1024ull * 2;
  const size_t size = bytes / sizeof(read_t);
  const int dimGrid = 512;
  const int dimBlock = 512;
  const int n = size / stride; // number of reads

  // allocate on gpu0 and enable peer access
  OR_SKIP_AND_RETURN(hipSetDevice(gpu0), NAME "failed to set device");
  OR_SKIP_AND_RETURN(hipMalloc(&src, bytes),
                     NAME " failed to perform hipMalloc");
  OR_SKIP_AND_RETURN(hipMemset(src, 0, bytes),
                     NAME " failed to perform src hipMemset");
  if (gpu0 != gpu1) {
    hipError_t err = hipDeviceEnablePeerAccess(gpu1, 0);
    if (hipSuccess != err && hipErrorPeerAccessAlreadyEnabled != err) {
      state.SkipWithError(NAME " failed to ensure peer access");
    }
  }

  // enable peer access from gpu1
  OR_SKIP_AND_RETURN(hipSetDevice(gpu1), NAME "failed to set device");
  if (gpu0 != gpu1) {
    hipError_t err = hipDeviceEnablePeerAccess(gpu0, 0);
    if (hipSuccess != err && hipErrorPeerAccessAlreadyEnabled != err) {
      state.SkipWithError(NAME " failed to ensure peer access");
    }
  }

  // run pull kernel on dst device
  OR_SKIP_AND_RETURN(hipSetDevice(gpu1), NAME " unable to set pull device");

  for (auto _ : state) {
    // Start copy
    OR_SKIP_AND_BREAK(hipEventRecord(start, stream),
                      NAME " failed to record start event");

    Comm_stride_pull_kernel<<<dimGrid, dimBlock, 0, stream>>>(src, n, stride,
                                                              nullptr);
    OR_SKIP_AND_BREAK(hipEventRecord(stop, stream),
                      NAME " failed to record stop event");

    // Wait for all copies to finish
    OR_SKIP_AND_BREAK(hipEventSynchronize(stop),
                      NAME " failed to synchronize");

    // Get the transfer time
    float millis;
    OR_SKIP_AND_BREAK(hipEventElapsedTime(&millis, start, stop),
                      NAME " failed to compute elapsed tiume");
    state.SetIterationTime(millis / 1000);
  }

  state.SetBytesProcessed(int64_t(state.iterations()) * n * sizeof(read_t));
  state.counters["ld-bytes"] = n * sizeof(read_t);
  state.counters["ld-count"] = n;
  state.counters["ld-size"] = sizeof(read_t);
  state.counters["gpu0"] = gpu0;
  state.counters["gpu1"] = gpu1;
  state.counters["alloc"] = bytes;
  state.counters["ld-stride"] = stride * sizeof(read_t);

  OR_SKIP_AND_RETURN(hipEventDestroy(start), "hipEventDestroy");
  OR_SKIP_AND_RETURN(hipEventDestroy(stop), "hipEventDestroy");
  OR_SKIP_AND_RETURN(hipStreamDestroy(stream), "hipStreamDestroy");
  OR_SKIP_AND_RETURN(hipFree(src), "hipFree");

#if defined(SCOPE_USE_NVTX)
  nvtxRangePop();
#endif
};

static void registerer() {
  std::string name;
  const std::vector<Device> cudas = scope::system::cuda_devices();
  for (size_t i = 0; i < cudas.size(); ++i) {
    for (size_t j = i; j < cudas.size(); ++j) {
      auto gpu0 = cudas[i];
      auto gpu1 = cudas[j];
      int ok1, ok2;
      if (!PRINT_IF_ERROR(hipDeviceCanAccessPeer(&ok1, gpu0, gpu1)) &&
          !PRINT_IF_ERROR(hipDeviceCanAccessPeer(&ok2, gpu1, gpu0))) {
        if ((ok1 && ok2) || i == j) {
          name = std::string(NAME) + "/" + std::to_string(gpu0) + "/" +
                 std::to_string(gpu1);
          benchmark::RegisterBenchmark(name.c_str(), Comm_stride_pull, gpu0,
                                       gpu1)
              ->STRIDE_ARGS()
              ->UseManualTime();
        }
      }
    }
  }
}

SCOPE_AFTER_INIT(registerer, NAME);
