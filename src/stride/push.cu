#include "hip/hip_runtime.h"
#include <sstream>

#include "scope/scope.hpp"

#include "args.hpp"

#define NAME "Comm_stride_push"

typedef int write_t;

static __global__ void Comm_stride_push_kernel(write_t *dst,
                                               const int n, // number of reads
                                               const int stride) {

  for (int i = blockDim.x * blockIdx.x + threadIdx.x; i < n;
       i += gridDim.x * blockDim.x) {
    dst[i * stride] = i;
  }
}

auto Comm_stride_push = [](benchmark::State &state, const int gpu0,
                           const int gpu1) {
  const int stride = state.range(0);

#if defined(SCOPE_USE_NVTX)
  {
    std::stringstream name;
    name << NAME << "/" << gpu0 << "/" << gpu1 << "/" << state.range(0) << "/"
         << state.range(1) << "/" << state.range(2);
    nvtxRangePush(name.str().c_str());
  }
#endif

  OR_SKIP_AND_RETURN(cuda_reset_device(gpu0),
                     NAME " failed to reset CUDA device");
  OR_SKIP_AND_RETURN(cuda_reset_device(gpu1),
                     NAME " failed to reset CUDA device");

  // fixed number of loads regardless of stride
  write_t *dst = nullptr;
  const size_t bytes = 1024ull * 1024ull * 1024ull * 2;
  const size_t size = bytes / sizeof(write_t);
  const int dimGrid = 512;
  const int dimBlock = 512;
  const int n = size / stride; // number of reads

  // gpu0 resources and  peer access
  OR_SKIP_AND_RETURN(hipSetDevice(gpu0), NAME "failed to set device");

  // create stream on src gpu (push)
  hipStream_t stream = nullptr;
  OR_SKIP_AND_RETURN(hipStreamCreate(&stream), NAME "failed to create stream");

  // Start and stop events on dst gpu (push)
  hipEvent_t start = nullptr;
  hipEvent_t stop = nullptr;
  OR_SKIP_AND_RETURN(hipEventCreate(&start), NAME " failed to create event");
  OR_SKIP_AND_RETURN(hipEventCreate(&stop), NAME " failed to create event");

  if (gpu0 != gpu1) {
    hipError_t err = hipDeviceEnablePeerAccess(gpu1, 0);
    if (hipSuccess != err && hipErrorPeerAccessAlreadyEnabled != err) {
      state.SkipWithError(NAME " failed to ensure peer access");
    }
  }



  // enable peer access from gpu1
  OR_SKIP_AND_RETURN(hipSetDevice(gpu1), NAME "failed to set device");
  OR_SKIP_AND_RETURN(hipMalloc(&dst, bytes),
                     NAME " failed to perform hipMalloc");
  OR_SKIP_AND_RETURN(hipMemset(dst, 0, bytes),
                     NAME " failed to perform dst hipMemset");
  if (gpu0 != gpu1) {
    hipError_t err = hipDeviceEnablePeerAccess(gpu0, 0);
    if (hipSuccess != err && hipErrorPeerAccessAlreadyEnabled != err) {
      state.SkipWithError(NAME " failed to ensure peer access");
    }
  }

  // run push kernel on src device
  OR_SKIP_AND_RETURN(hipSetDevice(gpu0), NAME " unable to set push device");

  for (auto _ : state) {
    // Start copy
    OR_SKIP_AND_BREAK(hipEventRecord(start, stream),
                      NAME " failed to record start event");

    Comm_stride_push_kernel<<<dimGrid, dimBlock, 0, stream>>>(dst, n, stride);
    OR_SKIP_AND_BREAK(hipEventRecord(stop, stream),
                      NAME " failed to record stop event");

    // Wait for all copies to finish
    OR_SKIP_AND_BREAK(hipEventSynchronize(stop),
                      NAME " failed to synchronize");

    // Get the transfer time
    float millis;
    OR_SKIP_AND_BREAK(hipEventElapsedTime(&millis, start, stop),
                      NAME " failed to compute elapsed tiume");
    state.SetIterationTime(millis / 1000);
  }

  state.SetBytesProcessed(int64_t(state.iterations()) * n * sizeof(write_t));
  state.counters["st-bytes"] = n * sizeof(write_t);
  state.counters["st-count"] = n;
  state.counters["st-size"] = sizeof(write_t);
  state.counters["gpu0"] = gpu0;
  state.counters["gpu1"] = gpu1;
  state.counters["alloc"] = bytes;
  state.counters["st-stride"] = stride * sizeof(write_t);

  OR_SKIP_AND_RETURN(hipEventDestroy(start), "hipEventDestroy");
  OR_SKIP_AND_RETURN(hipEventDestroy(stop), "hipEventDestroy");
  OR_SKIP_AND_RETURN(hipStreamDestroy(stream), "hipStreamDestroy");
  OR_SKIP_AND_RETURN(hipFree(dst), "hipFree");

#if defined(SCOPE_USE_NVTX)
  nvtxRangePop();
#endif
};

static void registerer() {
  std::string name;
  const std::vector<Device> cudas = scope::system::cuda_devices();
  for (size_t i = 0; i < cudas.size(); ++i) {
    for (size_t j = i; j < cudas.size(); ++j) {
      auto gpu0 = cudas[i];
      auto gpu1 = cudas[j];
      int ok1, ok2;
      if (!PRINT_IF_ERROR(hipDeviceCanAccessPeer(&ok1, gpu0, gpu1)) &&
          !PRINT_IF_ERROR(hipDeviceCanAccessPeer(&ok2, gpu1, gpu0))) {
        if ((ok1 && ok2) || i == j) {
          name = std::string(NAME) + "/" + std::to_string(gpu0) + "/" +
                 std::to_string(gpu1);
          benchmark::RegisterBenchmark(name.c_str(), Comm_stride_push, gpu0,
                                       gpu1)
              ->STRIDE_ARGS()
              ->UseManualTime();
        }
      }
    }
  }
}

SCOPE_AFTER_INIT(registerer, NAME);
