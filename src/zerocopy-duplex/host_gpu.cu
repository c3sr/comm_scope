#include "hip/hip_runtime.h"
/* Use half the GPU to write to the host and the other half to read from the
 * host
 */

#include "scope/scope.hpp"

#include "args.hpp"
#include "kernels.hu"

constexpr int rdDimBlock = 256;
constexpr int wrDimBlock = 256;
auto GpuWrFunc = gpu_write<rdDimBlock, int32_t>;
auto GpuRdFunc = gpu_read<rdDimBlock, int32_t>;

#define NAME Comm_ZeroCopy_Duplex_HostGPU
#define NAME2 "Comm_ZeroCopy_Duplex_HostGPU"

namespace ns_NAME {
__global__ void busy_wait(clock_t *d, clock_t clock_count) {
  clock_t start_clock = clock64();
  clock_t clock_offset = 0;
  while (clock_offset < clock_count) {
    clock_offset = clock64() - start_clock;
  }
  if (d) {
    *d = clock_offset;
  }
}
} // namespace ns_NAME

auto Comm_ZeroCopy_Duplex_HostGPU = [](benchmark::State &state, const int numa,
                                       const int cuda) {
  using namespace ns_NAME;

  numa::ScopedBind binder(numa);

  const size_t pageSize = page_size();

  const auto bytes = 1ULL << static_cast<size_t>(state.range(0));

  hipStream_t stream[2];
  void *cpu[2] = {};
  hipEvent_t start[2] = {};
  hipEvent_t stop[2] = {};

  OR_SKIP_AND_RETURN(cuda_reset_device(cuda), "");

  cpu[0] = aligned_alloc(pageSize, bytes);
  cpu[1] = aligned_alloc(pageSize, bytes);
  defer(free(cpu[0]));
  defer(free(cpu[1]));
  if (bytes && (!cpu[0] || !cpu[1])) {
    state.SkipWithError(NAME2 " failed to allocate host memory");
    return;
  }
  std::memset(cpu[0], 0xDEADBEEF, bytes);
  std::memset(cpu[1], 0xDEADBEEF, bytes);

  OR_SKIP_AND_RETURN(hipHostRegister(cpu[0], bytes, hipHostRegisterMapped),
                     "");
  OR_SKIP_AND_RETURN(hipHostRegister(cpu[1], bytes, hipHostRegisterMapped),
                     "");
  defer(hipHostUnregister(cpu[0]));
  defer(hipHostUnregister(cpu[1]));

  // get a valid device pointers
  void *dptr[2] = {};
  hipDeviceProp_t prop;
  OR_SKIP_AND_RETURN(hipGetDeviceProperties(&prop, cuda), "");
#if __CUDACC_VER_MAJOR__ >= 9
  if (prop.canUseHostPointerForRegisteredMem) {
#else
  if (false) {
#endif
    dptr[0] = cpu[0];
    dptr[1] = cpu[1];
  } else {
    OR_SKIP_AND_RETURN(hipHostGetDevicePointer(&dptr[0], cpu[0], 0), "");
    OR_SKIP_AND_RETURN(hipHostGetDevicePointer(&dptr[1], cpu[1], 0), "");
  }

  // create streams
  OR_SKIP_AND_RETURN(hipStreamCreate(&stream[0]), "");
  OR_SKIP_AND_RETURN(hipStreamCreate(&stream[1]), "");
  defer(hipStreamDestroy(stream[0]));
  defer(hipStreamDestroy(stream[1]));

  OR_SKIP_AND_RETURN(hipEventCreate(&start[0]), "");
  OR_SKIP_AND_RETURN(hipEventCreate(&start[1]), "");
  defer(hipEventDestroy(start[0]));
  defer(hipEventDestroy(start[1]));
  OR_SKIP_AND_RETURN(hipEventCreate(&stop[0]), "");
  OR_SKIP_AND_RETURN(hipEventCreate(&stop[1]), "");
  defer(hipEventDestroy(stop[0]));
  defer(hipEventDestroy(stop[1]));

  // compute kernel params that are half of GPU
  int rdDimGrid;
  int wrDimGrid;
  {
    int maxActiveBlocks;
    hipOccupancyMaxActiveBlocksPerMultiprocessor(&maxActiveBlocks, GpuWrFunc,
                                                  wrDimBlock, 0);
    wrDimGrid = 0.5 * maxActiveBlocks * prop.multiProcessorCount;
  }
  {
    int maxActiveBlocks;
    hipOccupancyMaxActiveBlocksPerMultiprocessor(&maxActiveBlocks, GpuRdFunc,
                                                  rdDimBlock, 0);
    rdDimGrid = 0.5 * maxActiveBlocks * prop.multiProcessorCount;
  }

  clock_t cycles = 16384;
  for (auto _ : state) {
  restart_iteration:

    // launch the busy-wait kernel
    busy_wait<<<1, 1, 0, stream[0]>>>(nullptr, cycles);

    // set up the copies
    OR_SKIP_AND_BREAK(hipEventRecord(start[0], stream[0]),
                      ""); // stream 0 start
    gpu_read<rdDimBlock><<<rdDimGrid, rdDimBlock, 0, stream[0]>>>(
        (int32_t *)dptr[0], (int32_t *)nullptr, bytes); // stream 0 copy
    OR_SKIP_AND_BREAK(hipStreamWaitEvent(stream[1], start[0], 0),
                      ""); // stream 1 wait for stream 0 to start
    // OR_SKIP_AND_BREAK(hipEventRecord(start[1], stream[1]),
    //                   ""); // stream 1 start
    gpu_write<wrDimBlock><<<wrDimGrid, wrDimBlock, 0, stream[1]>>>(
        (int32_t *)dptr[1], bytes);                             // stream 1 copy
    OR_SKIP_AND_BREAK(hipEventRecord(stop[1], stream[1]), ""); // stream 1 stop
    OR_SKIP_AND_BREAK(hipStreamWaitEvent(stream[0], stop[1], 0),
                      ""); // stream 0 wait for stream 1 to stop
    OR_SKIP_AND_BREAK(hipEventRecord(stop[0], stream[0]), ""); // stream 0 stop

    // wait for streams to finish work, and restart iteration if needed
    hipError_t err = hipEventQuery(start[0]);
    if (hipSuccess == err) {
      // busy-wait is done, so it was too slow.
      OR_SKIP_AND_BREAK(hipStreamSynchronize(stream[0]), "");
      OR_SKIP_AND_BREAK(hipStreamSynchronize(stream[1]), "");
      cycles *= 2;
      goto restart_iteration;
    } else if (hipErrorNotReady == err) {
      // kernel was long enough
    } else {
      OR_SKIP_AND_BREAK(err, "errored while waiting for kernel");
    }

    OR_SKIP_AND_BREAK(hipEventSynchronize(stop[0]), "");
    float millis = 0;
    OR_SKIP_AND_BREAK(hipEventElapsedTime(&millis, start[0], stop[0]), "");
    state.SetIterationTime(millis / 1000);
  }

  state.SetBytesProcessed(int64_t(state.iterations()) * int64_t(bytes) * 2);
  state.counters["bytes"] = bytes;
  state.counters["numa"] = numa;
  state.counters["cuda"] = cuda;
  state.counters["cycles"] = cycles;
  state.counters["rd_blocks"] = wrDimGrid;
  state.counters["wr_blocks"] = rdDimGrid;
};

static void registerer() {

  for (int cuda : scope::system::cuda_devices()) {
    for (auto numa : numa::mems()) {

      std::string name(NAME2);
      name += "/" + std::to_string(numa) + "/" + std::to_string(cuda);
      benchmark::RegisterBenchmark(name.c_str(), Comm_ZeroCopy_Duplex_HostGPU,
                                   numa, cuda)
          ->ARGS()
          ->UseManualTime();
    }
  }
}

SCOPE_AFTER_INIT(registerer, NAME2);
